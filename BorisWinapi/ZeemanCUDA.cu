#include "hip/hip_runtime.h"
#include "ZeemanCUDA.h"

#if COMPILECUDA == 1

#ifdef MODULE_ZEEMAN

#include "BorisCUDALib.cuh"

#include "Mesh_FerromagneticCUDA.h"
#include "MeshParamsControlCUDA.h"

__global__ void ZeemanCUDA_UpdateField(ManagedMeshCUDA& cuMesh, cuReal3& Ha, cuBReal& energy, bool do_reduction)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC<cuReal3>& Heff = *cuMesh.pHeff;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal energy_ = 0.0;

	if (idx < Heff.linear_size()) {

		cuBReal cHA = *cuMesh.pcHA;
		cuMesh.update_parameters_mcoarse(idx, *cuMesh.pcHA, cHA);

		Heff[idx] += (cHA * Ha);

		if (do_reduction) {

			int non_empty_cells = M.get_nonempty_cells();
			if (non_empty_cells) energy_ = -(cuBReal)MU0 * M[idx] * (cHA * Ha) / non_empty_cells;
		}
	}

	if(do_reduction) reduction_sum(0, 1, &energy_, energy);
}

//----------------------- UpdateField LAUNCHER

void ZeemanCUDA::UpdateField(void)
{
	if (pMeshCUDA->CurrentTimeStepSolved()) {

		ZeroEnergy();

		ZeemanCUDA_UpdateField << < (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >> > (pMeshCUDA->cuMesh, Ha, energy, true);
	}
	else ZeemanCUDA_UpdateField << < (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >> > (pMeshCUDA->cuMesh, Ha, energy, false);
}

#endif

#endif