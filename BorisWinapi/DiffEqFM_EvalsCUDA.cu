#include "hip/hip_runtime.h"
#include "DiffEqFMCUDA.h"

#if COMPILECUDA == 1
#ifdef MESH_COMPILATION_FERROMAGNETIC

//defines evaluation methods kernel launchers

#include "BorisCUDALib.cuh"

//-----------------------------------------

__global__ void RestoreMagnetisation_FM_kernel(cuVEC_VC<cuReal3>& M, cuVEC<cuReal3>& sM1)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < M.linear_size()) {

		M[idx] = sM1[idx];
	}
}

//Restore magnetisation after a failed step for adaptive time-step methods
void DifferentialEquationFMCUDA::RestoreMagnetisation(void)
{
	RestoreMagnetisation_FM_kernel <<< (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> (pMeshCUDA->M, sM1);
}

//-----------------------------------------

#endif
#endif