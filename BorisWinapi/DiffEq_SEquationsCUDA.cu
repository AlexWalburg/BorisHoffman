#include "hip/hip_runtime.h"
#include "DiffEqCUDA.h"

#if COMPILECUDA == 1

#include "BorisCUDALib.h"
#include "BorisCUDALib.cuh"

//---------------------------------------- OTHER CALCULATION METHODS : GENERATE THERMAL cuVECs

//----------------------------------------

__global__ void GenerateThermalField_Kernel(cuBorisRand& prng, ManagedDiffEqCUDA& cuDiffEq, ManagedMeshCUDA& cuMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuReal grel = cuMesh.pgrel->get0();

	if (idx < cuDiffEq.pH_Thermal->linear_size() && cuIsNZ(grel)) {

		if (cuMesh.pM->is_not_empty(idx) && !cuMesh.pM->is_skipcell(idx)) {

			cuReal3 h = cuMesh.pM->h;
			cuReal dT = *cuDiffEq.pdT;

			cuReal Temperature;

			if (cuMesh.pTemp->linear_size()) {

				//get temperature at centre of idx M cell
				Temperature = (*cuMesh.pTemp)[cuMesh.pM->cellidx_to_position(idx)];
			}
			else Temperature = (*cuMesh.pbase_temperature);

			//do not include any damping here - this will be included in the stochastic equations
			cuReal Mag = prng.rand() * sqrt(2 * (cuReal)BOLTZMANN * Temperature / ((cuReal)GAMMA * grel * h.dim() * (cuReal)MU0 * cuMesh.pMs->get0() * dT));
			cuReal theta = prng.rand() * (cuReal)TWO_PI;
			cuReal phi = prng.rand() * (cuReal)TWO_PI;

			(*cuDiffEq.pH_Thermal)[idx] = Mag * cuReal3(cos(phi)*sin(theta), sin(phi)*sin(theta), cos(theta));
		}
	}
}

//called when using stochastic equations
void DifferentialEquationCUDA::GenerateThermalField(void)
{
	GenerateThermalField_Kernel <<< (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> (prng, cuDiffEq, pMeshCUDA->cuMesh);
}

//----------------------------------------

__global__ void GenerateThermalField_and_Torque_Kernel(cuBorisRand& prng, ManagedDiffEqCUDA& cuDiffEq, ManagedMeshCUDA& cuMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuReal grel = cuMesh.pgrel->get0();

	if (idx < cuDiffEq.pH_Thermal->linear_size() && cuIsNZ(grel)) {

		if (cuMesh.pM->is_not_empty(idx) && !cuMesh.pM->is_skipcell(idx)) {

			cuReal3 h = cuMesh.pM->h;
			cuReal dT = *cuDiffEq.pdT;

			cuReal Temperature;

			if (cuMesh.pTemp->linear_size()) {
				
				//get temperature at centre of idx M cell
				Temperature = (*cuMesh.pTemp)[cuMesh.pM->cellidx_to_position(idx)];
			}
			else Temperature = (*cuMesh.pbase_temperature);

			//1. Thermal Field
			//do not include any damping here - this will be included in the stochastic equations
			cuReal Mag = prng.rand() * sqrt(2 * (cuReal)BOLTZMANN * Temperature / ((cuReal)GAMMA * grel * h.dim() * (cuReal)MU0 * cuMesh.pMs->get0() * dT));
			cuReal theta = prng.rand() * (cuReal)TWO_PI;
			cuReal phi = prng.rand() * (cuReal)TWO_PI;

			(*cuDiffEq.pH_Thermal)[idx] = Mag * cuReal3(cos(phi)*sin(theta), sin(phi)*sin(theta), cos(theta));

			//2. Thermal Torque
			//do not include any damping here - this will be included in the stochastic equations
			Mag = prng.rand() * sqrt(2 * (cuReal)BOLTZMANN * Temperature * (cuReal)GAMMA * grel * cuMesh.pMs->get0() / ((cuReal)MU0 * h.dim() * dT));
			theta = prng.rand() * (cuReal)TWO_PI;
			phi = prng.rand() * (cuReal)TWO_PI;

			(*cuDiffEq.pTorque_Thermal)[idx] = Mag * cuReal3(cos(phi)*sin(theta), sin(phi)*sin(theta), cos(theta));
		}
	}
}

void DifferentialEquationCUDA::GenerateThermalField_and_Torque(void)
{
	GenerateThermalField_and_Torque_Kernel <<< (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> (prng, cuDiffEq, pMeshCUDA->cuMesh);
}

#endif