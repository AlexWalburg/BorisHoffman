#include "hip/hip_runtime.h"
#include "DMExchangeCUDA.h"

#if COMPILECUDA == 1

#ifdef MODULE_DMEXCHANGE

#include "BorisCUDALib.cuh"

#include "Mesh_FerromagneticCUDA.h"
#include "MeshParamsControlCUDA.h"

__global__ void DMExchangeCUDA_UpdateField(ManagedMeshCUDA& cuMesh, cuBReal& energy, bool do_reduction)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC<cuReal3>& Heff = *cuMesh.pHeff;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal energy_ = 0.0;

	if (idx < Heff.linear_size()) {

		cuReal3 Hexch = cuReal3();

		if (M.is_not_empty(idx)) {

			cuBReal Ms = *cuMesh.pMs;
			cuBReal A = *cuMesh.pA;
			cuBReal D = *cuMesh.pD;
			cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs, Ms, *cuMesh.pA, A, *cuMesh.pD, D);

			if (M.is_interior(idx)) {

				//interior point : can use cheaper neu versions

				//direct exchange contribution
				Hexch = 2 * A * M.delsq_neu(idx) / ((cuBReal)MU0 * Ms * Ms);

				//Dzyaloshinskii-Moriya exchange contribution

				//Hdm, ex = -2D / (mu0*Ms) * curl m
				Hexch += -2 * D * M.curl_neu(idx) / ((cuBReal)MU0 * Ms * Ms);
			}
			else {

				//Non-homogeneous Neumann boundary conditions apply when using DMI. Required to ensure Brown's condition is fulfilled, i.e. m x h -> 0 when relaxing.
				cuReal3 bnd_dm_dx = (D / (2 * A)) * cuReal3(0, -M[idx].z, M[idx].y);
				cuReal3 bnd_dm_dy = (D / (2 * A)) * cuReal3(M[idx].z, 0, -M[idx].x);
				cuReal3 bnd_dm_dz = (D / (2 * A)) * cuReal3(-M[idx].y, M[idx].x, 0);
				cuReal33 bnd_nneu = cuReal33(bnd_dm_dx, bnd_dm_dy, bnd_dm_dz);

				//direct exchange contribution
				Hexch = 2 * A * M.delsq_nneu(idx, bnd_nneu) / ((cuBReal)MU0 * Ms * Ms);

				//Dzyaloshinskii-Moriya exchange contribution

				//Hdm, ex = -2D / (mu0*Ms) * curl m
				Hexch += -2 * D * M.curl_nneu(idx, bnd_nneu) / ((cuBReal)MU0 * Ms * Ms);
			}

			if (do_reduction) {

				int non_empty_cells = M.get_nonempty_cells();
				if (non_empty_cells) energy_ = -(cuBReal)MU0 * M[idx] * Hexch / (2 * non_empty_cells);
			}
		}

		Heff[idx] += Hexch;
	}

	if (do_reduction) reduction_sum(0, 1, &energy_, energy);
}

//----------------------- UpdateField LAUNCHER

void DMExchangeCUDA::UpdateField(void)
{
	if (pMeshCUDA->CurrentTimeStepSolved()) {

		ZeroEnergy();

		DMExchangeCUDA_UpdateField << < (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >> > (pMeshCUDA->cuMesh, energy, true);
	}
	else {

		DMExchangeCUDA_UpdateField << < (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >> > (pMeshCUDA->cuMesh, energy, false);
	}

	if (pMeshCUDA->GetMeshExchangeCoupling()) CalculateExchangeCoupling(energy);
}

#endif

#endif