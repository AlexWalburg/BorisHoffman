#include "hip/hip_runtime.h"
#include "DMExchangeCUDA.h"

#if COMPILECUDA == 1

#ifdef MODULE_DMEXCHANGE

#include "BorisCUDALib.cuh"

#include "Mesh_FerromagneticCUDA.h"
#include "MeshParamsControlCUDA.h"

__global__ void DMExchangeCUDA_UpdateField(ManagedMeshCUDA& cuMesh, cuReal& energy, bool do_reduction)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC<cuReal3>& Heff = *cuMesh.pHeff;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuReal energy_ = 0.0;

	if (idx < Heff.linear_size()) {

		cuReal3 Hexch = cuReal3();

		if (M.is_not_empty(idx)) {

			cuReal Ms = *cuMesh.pMs;
			cuReal A = *cuMesh.pA;
			cuReal D = *cuMesh.pD;
			cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs, Ms, *cuMesh.pA, A, *cuMesh.pD, D);

			//direct exchange contribution
			Hexch = 2 * A * M.delsq_neu(idx) / ((cuReal)MU0 * Ms * Ms);

			//Dzyaloshinskii-Moriya exchange contribution

			//Hdm, ex = -2D / (mu0*Ms) * curl m
			Hexch += -2 * D * M.curl_neu(idx) / ((cuReal)MU0 * Ms * Ms);

			if (do_reduction) {

				int non_empty_cells = M.get_nonempty_cells();
				if (non_empty_cells) energy_ = -(cuReal)MU0 * M[idx] * Hexch / (2 * non_empty_cells);
			}
		}

		Heff[idx] += Hexch;
	}

	if (do_reduction) reduction_sum(0, 1, &energy_, energy);
}

//----------------------- UpdateField LAUNCHER

void DMExchangeCUDA::UpdateField(void)
{
	if (pMeshCUDA->CurrentTimeStepSolved()) {

		ZeroEnergy();

		DMExchangeCUDA_UpdateField << < (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >> > (pMeshCUDA->cuMesh, energy, true);
	}
	else {

		DMExchangeCUDA_UpdateField << < (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >> > (pMeshCUDA->cuMesh, energy, false);
	}
}

#endif

#endif