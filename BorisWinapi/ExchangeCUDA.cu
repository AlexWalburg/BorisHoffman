#include "hip/hip_runtime.h"
#include "ExchangeCUDA.h"

#if COMPILECUDA == 1

#ifdef MODULE_EXCHANGE

#include "BorisCUDALib.cuh"

#include "MeshCUDA.h"
#include "MeshParamsControlCUDA.h"
#include "MeshDefs.h"

__global__ void ExchangeCUDA_FM_UpdateField(ManagedMeshCUDA& cuMesh, cuBReal& energy, bool do_reduction)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC<cuReal3>& Heff = *cuMesh.pHeff;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal energy_ = 0.0;

	if (idx < Heff.linear_size()) {

		cuReal3 Hexch = cuReal3();

		if (M.is_not_empty(idx)) {

			cuBReal Ms = *cuMesh.pMs;
			cuBReal A = *cuMesh.pA;
			cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs, Ms, *cuMesh.pA, A);

			Hexch = 2 * A * M.delsq_neu(idx) / ((cuBReal)MU0 * Ms * Ms);
			
			if (do_reduction) {

				int non_empty_cells = M.get_nonempty_cells();
				if (non_empty_cells) energy_ = -(cuBReal)MU0 * M[idx] * Hexch / (2 * non_empty_cells);
			}
		}

		Heff[idx] += Hexch;
	}

	if (do_reduction) reduction_sum(0, 1, &energy_, energy);
}

__global__ void ExchangeCUDA_AFM_UpdateField(ManagedMeshCUDA& cuMesh, cuBReal& energy, bool do_reduction)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC<cuReal3>& Heff = *cuMesh.pHeff;

	cuVEC_VC<cuReal3>& M2 = *cuMesh.pM2;
	cuVEC<cuReal3>& Heff2 = *cuMesh.pHeff2;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal energy_ = 0.0;

	if (idx < Heff.linear_size()) {

		cuReal3 Hexch = cuReal3();
		cuReal3 Hexch2 = cuReal3();

		if (M.is_not_empty(idx)) {

			cuReal2 Ms_AFM = *cuMesh.pMs_AFM;
			cuReal2 A_AFM = *cuMesh.pA_AFM;
			cuBReal A12 = *cuMesh.pA12;

			cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs_AFM, Ms_AFM, *cuMesh.pA_AFM, A_AFM, *cuMesh.pA12, A12);

			Hexch = 2 * A_AFM.i * M.delsq_neu(idx) / ((cuBReal)MU0 * Ms_AFM.i * Ms_AFM.i) + (4 * A12 / (MU0*Ms_AFM.i*Ms_AFM.j)) * M2[idx];
			Hexch2 = 2 * A_AFM.j * M2.delsq_neu(idx) / ((cuBReal)MU0 * Ms_AFM.j * Ms_AFM.j) + (4 * A12 / (MU0*Ms_AFM.i*Ms_AFM.j)) * M[idx];

			if (do_reduction) {

				int non_empty_cells = M.get_nonempty_cells();
				if (non_empty_cells) energy_ = -(cuBReal)MU0 * (M[idx] * Hexch  + M2[idx] * Hexch2) / (4 * non_empty_cells);
			}
		}

		Heff[idx] += Hexch;
		Heff2[idx] += Hexch2;
	}

	if (do_reduction) reduction_sum(0, 1, &energy_, energy);
}

//----------------------- UpdateField LAUNCHER

void Exch_6ngbr_NeuCUDA::UpdateField(void)
{
	if (pMeshCUDA->GetMeshType() == MESH_ANTIFERROMAGNETIC) {

		//anti-ferromagnetic mesh

		if (pMeshCUDA->CurrentTimeStepSolved()) {

			ZeroEnergy();

			ExchangeCUDA_AFM_UpdateField << < (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >> > (pMeshCUDA->cuMesh, energy, true);
		}
		else {

			ExchangeCUDA_AFM_UpdateField << < (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >> > (pMeshCUDA->cuMesh, energy, false);
		}
	}
	else {

		//ferromagnetic mesh

		if (pMeshCUDA->CurrentTimeStepSolved()) {

			ZeroEnergy();

			ExchangeCUDA_FM_UpdateField << < (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >> > (pMeshCUDA->cuMesh, energy, true);
		}
		else {

			ExchangeCUDA_FM_UpdateField << < (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >> > (pMeshCUDA->cuMesh, energy, false);
		}
	}
	
	if (pMeshCUDA->GetMeshExchangeCoupling()) CalculateExchangeCoupling(energy);
}

#endif

#endif