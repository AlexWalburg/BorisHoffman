#include "hip/hip_runtime.h"
#include "DiffEqCUDA.h"
#include "MeshParamsControlCUDA.h"

#if COMPILECUDA == 1

//defines evaluation methods kernel launchers

#include "BorisCUDALib.cuh"

//----------------------------------------- AUXILIARY

__global__ void Zerovalues_kernel(cuReal& mxh, cuReal3& mxh_av, size_t& avpoints, cuReal& dmdt, cuReal3& dmdt_av, size_t& avpoints2, cuReal& lte)
{
	if (threadIdx.x == 0) mxh = 0.0;
	else if (threadIdx.x == 1) mxh_av = cuReal3(0.0);
	else if (threadIdx.x == 2) avpoints = 0;
	else if (threadIdx.x == 3) dmdt = 0.0;
	else if (threadIdx.x == 4) dmdt_av = cuReal3(0.0);
	else if (threadIdx.x == 5) avpoints2 = 0;
	else if (threadIdx.x == 6) lte = 0.0;
}

void ODECommonCUDA::Zero_reduction_values(void)
{
	Zerovalues_kernel <<< 1, CUDATHREADS >>> (*pmxh, *pmxh_av, *pavpoints, *pdmdt, *pdmdt_av, *pavpoints2, *plte);
}

//-----------------------------------------

__global__ void Zeromxh_kernel(cuReal& mxh, cuReal3& mxh_av, size_t& avpoints, cuReal& lte)
{
	if (threadIdx.x == 0) mxh = 0.0;
	else if (threadIdx.x == 1) mxh_av = cuReal3(0.0);
	else if (threadIdx.x == 2) avpoints = 0;
	else if (threadIdx.x == 3) lte = 0.0;
}

void ODECommonCUDA::Zero_mxh_lte_values(void)
{
	Zeromxh_kernel << < 1, CUDATHREADS >> > (*pmxh, *pmxh_av, *pavpoints, *plte);
}

//-----------------------------------------

__global__ void Zerodmdt_kernel(cuReal& dmdt, cuReal3& dmdt_av, size_t& avpoints2, cuReal& lte)
{
	if (threadIdx.x == 0) dmdt = 0.0;
	else if (threadIdx.x == 1) dmdt_av = cuReal3(0.0);
	else if (threadIdx.x == 2) avpoints2 = 0;
	else if (threadIdx.x == 3) lte = 0.0;
}

void ODECommonCUDA::Zero_dmdt_lte_values(void)
{
	Zerodmdt_kernel << < 1, CUDATHREADS >> > (*pdmdt, *pdmdt_av, *pavpoints2, *plte);
}

//-----------------------------------------

__global__ void Zerolte_kernel(cuReal& lte)
{
	if (threadIdx.x == 0) lte = 0.0;
}

void ODECommonCUDA::Zero_lte_value(void)
{
	Zerolte_kernel << < 1, CUDATHREADS >> > (*plte);
}

//-----------------------------------------

__global__ void mxhav_to_mxh_kernel(cuReal& mxh, cuReal3& mxh_av, size_t& avpoints)
{
	if (threadIdx.x == 0) {

		if (avpoints) {

			mxh = cu_GetMagnitude(mxh_av) / avpoints;
		}
		else {

			mxh = 0.0;
		}
	}
}

void ODECommonCUDA::mxhav_to_mxh(void)
{
	mxhav_to_mxh_kernel <<< 1, CUDATHREADS >>> (*pmxh, *pmxh_av, *pavpoints);
}

//-----------------------------------------

__global__ void dmdtav_to_dmdt_kernel(cuReal& dmdt, cuReal3& dmdt_av, size_t& avpoints2)
{
	if (threadIdx.x == 0) {

		if (avpoints2) {

			dmdt = cu_GetMagnitude(dmdt_av) / avpoints2;
		}
		else {

			dmdt = 0.0;
		}
	}
}

void ODECommonCUDA::dmdtav_to_dmdt(void)
{
	dmdtav_to_dmdt_kernel << < 1, CUDATHREADS >> > (*pdmdt, *pdmdt_av, *pavpoints2);
}

//-----------------------------------------

__global__ void Zero_SD_Solver_BB_Values_kernel(cuReal& delta_M_sq, cuReal& delta_G_sq, cuReal& delta_M_dot_delta_G)
{
	if (threadIdx.x == 0) delta_M_sq = 0.0;
	else if (threadIdx.x == 1) delta_G_sq = 0.0;
	else if (threadIdx.x == 2) delta_M_dot_delta_G = 0.0;
}

void ODECommonCUDA::Zero_SD_Solver_BB_Values(void)
{
	Zero_SD_Solver_BB_Values_kernel <<< 1, CUDATHREADS >>> (*pdelta_M_sq, *pdelta_G_sq, *pdelta_M_dot_delta_G);
}

//-----------------------------------------

__global__ void RestoreMagnetisation_kernel(cuVEC_VC<cuReal3>& M, cuVEC<cuReal3>& sM1)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < M.linear_size()) {

		M[idx] = sM1[idx];
	}
}

//Restore magnetisation after a failed step for adaptive time-step methods
void DifferentialEquationCUDA::RestoreMagnetisation(void)
{
	RestoreMagnetisation_kernel <<< (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> (pMeshCUDA->M, sM1);
}

//-----------------------------------------

#endif