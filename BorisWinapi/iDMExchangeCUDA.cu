#include "hip/hip_runtime.h"
#include "iDMExchangeCUDA.h"

#if COMPILECUDA == 1

#ifdef MODULE_IDMEXCHANGE

#include "BorisCUDALib.cuh"

#include "Mesh_FerromagneticCUDA.h"
#include "MeshParamsControlCUDA.h"

__global__ void iDMExchangeCUDA_UpdateField(ManagedMeshCUDA& cuMesh, cuReal& energy, bool do_reduction)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC<cuReal3>& Heff = *cuMesh.pHeff;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuReal energy_ = 0.0;

	if (idx < Heff.linear_size()) {

		cuReal3 Hexch = cuReal3();

		if (M.is_not_empty(idx)) {

			cuReal Ms = *cuMesh.pMs;
			cuReal A = *cuMesh.pA;
			cuReal D = *cuMesh.pD;
			cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs, Ms, *cuMesh.pA, A, *cuMesh.pD, D);

			if (M.is_plane_interior(idx)) {

				//interior point : can use cheaper neu versions

				//direct exchange contribution
				Hexch = 2 * A * M.delsq_neu(idx) / ((cuReal)MU0 * Ms * Ms);

				//Dzyaloshinskii-Moriya interfacial exchange contribution

				//Differentials of M components (we only need 4, not all 9 so this could be optimised). First index is the differential direction, second index is the M component
				cuReal33 Mdiff = M.grad_neu(idx);

				//Hdm, ex = -2D / (mu0*Ms) * (dmz / dx, dmz / dy, -dmx / dx - dmy / dy)
				Hexch += -2 * D * cuReal3(Mdiff.x.z, Mdiff.y.z, -Mdiff.x.x - Mdiff.y.y) / ((cuReal)MU0 * Ms * Ms);
			}
			else {

				//Non-homogeneous Neumann boundary conditions apply when using DMI. Required to ensure Brown's condition is fulfilled, i.e. m x h -> 0 when relaxing.
				cuReal3 bnd_dm_dx = (D / (2 * A * Ms)) * cuReal3(M[idx].z, 0, -M[idx].x);
				cuReal3 bnd_dm_dy = (D / (2 * A * Ms)) * cuReal3(0, M[idx].z, -M[idx].y);
				cuReal33 bnd_nneu = cuReal33(bnd_dm_dx, bnd_dm_dy, cuReal3());

				//direct exchange contribution
				Hexch = 2 * A * M.delsq_nneu(idx, bnd_nneu) / ((cuReal)MU0 * Ms * Ms);

				//Dzyaloshinskii-Moriya interfacial exchange contribution

				//Differentials of M components (we only need 4, not all 9 so this could be optimised). First index is the differential direction, second index is the M component
				cuReal33 Mdiff = M.grad_nneu(idx, bnd_nneu);

				//Hdm, ex = -2D / (mu0*Ms) * (dmz / dx, dmz / dy, -dmx / dx - dmy / dy)
				Hexch += -2 * D * cuReal3(Mdiff.x.z, Mdiff.y.z, -Mdiff.x.x - Mdiff.y.y) / ((cuReal)MU0 * Ms * Ms);
			}

			if (do_reduction) {

				int non_empty_cells = M.get_nonempty_cells();
				if (non_empty_cells) energy_ = -(cuReal)MU0 * M[idx] * Hexch / (2 * non_empty_cells);
			}
		}

		Heff[idx] += Hexch;
	}

	if (do_reduction) reduction_sum(0, 1, &energy_, energy);
}

//----------------------- UpdateField LAUNCHER

void iDMExchangeCUDA::UpdateField(void)
{
	if (pMeshCUDA->CurrentTimeStepSolved()) {

		ZeroEnergy();

		iDMExchangeCUDA_UpdateField << < (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >> > (pMeshCUDA->cuMesh, energy, true);
	}
	else {

		iDMExchangeCUDA_UpdateField << < (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >> > (pMeshCUDA->cuMesh, energy, false);
	}

	if (pMeshCUDA->GetMeshExchangeCoupling()) CalculateExchangeCoupling(energy);
}

#endif

#endif