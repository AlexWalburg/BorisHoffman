#include "hip/hip_runtime.h"
#include "iDMExchangeCUDA.h"

#if COMPILECUDA == 1

#ifdef MODULE_IDMEXCHANGE

#include "BorisCUDALib.cuh"

#include "MeshCUDA.h"
#include "MeshParamsControlCUDA.h"
#include "MeshDefs.h"

__global__ void iDMExchangeCUDA_FM_UpdateField(ManagedMeshCUDA& cuMesh, cuBReal& energy, bool do_reduction)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC<cuReal3>& Heff = *cuMesh.pHeff;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal energy_ = 0.0;

	if (idx < Heff.linear_size()) {

		cuReal3 Hexch = cuReal3();

		if (M.is_not_empty(idx)) {

			cuBReal Ms = *cuMesh.pMs;
			cuBReal A = *cuMesh.pA;
			cuBReal D = *cuMesh.pD;
			cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs, Ms, *cuMesh.pA, A, *cuMesh.pD, D);

			if (M.is_plane_interior(idx)) {

				//interior point : can use cheaper neu versions

				//direct exchange contribution
				Hexch = 2 * A * M.delsq_neu(idx) / ((cuBReal)MU0 * Ms * Ms);

				//Dzyaloshinskii-Moriya interfacial exchange contribution

				//Differentials of M components (we only need 4, not all 9 so this could be optimised). First index is the differential direction, second index is the M component
				cuReal33 Mdiff = M.grad_neu(idx);

				//Hdm, ex = -2D / (mu0*Ms) * (dmz / dx, dmz / dy, -dmx / dx - dmy / dy)
				Hexch += -2 * D * cuReal3(Mdiff.x.z, Mdiff.y.z, -Mdiff.x.x - Mdiff.y.y) / ((cuBReal)MU0 * Ms * Ms);
			}
			else {

				//Non-homogeneous Neumann boundary conditions apply when using DMI. Required to ensure Brown's condition is fulfilled, i.e. m x h -> 0 when relaxing.
				cuReal3 bnd_dm_dx = (D / (2 * A)) * cuReal3(M[idx].z, 0, -M[idx].x);
				cuReal3 bnd_dm_dy = (D / (2 * A)) * cuReal3(0, M[idx].z, -M[idx].y);
				cuReal33 bnd_nneu = cuReal33(bnd_dm_dx, bnd_dm_dy, cuReal3());

				//direct exchange contribution
				Hexch = 2 * A * M.delsq_nneu(idx, bnd_nneu) / ((cuBReal)MU0 * Ms * Ms);

				//Dzyaloshinskii-Moriya interfacial exchange contribution

				//Differentials of M components (we only need 4, not all 9 so this could be optimised). First index is the differential direction, second index is the M component
				cuReal33 Mdiff = M.grad_nneu(idx, bnd_nneu);

				//Hdm, ex = -2D / (mu0*Ms) * (dmz / dx, dmz / dy, -dmx / dx - dmy / dy)
				Hexch += -2 * D * cuReal3(Mdiff.x.z, Mdiff.y.z, -Mdiff.x.x - Mdiff.y.y) / ((cuBReal)MU0 * Ms * Ms);
			}

			if (do_reduction) {

				int non_empty_cells = M.get_nonempty_cells();
				if (non_empty_cells) energy_ = -(cuBReal)MU0 * M[idx] * Hexch / (2 * non_empty_cells);
			}
		}

		Heff[idx] += Hexch;
	}

	if (do_reduction) reduction_sum(0, 1, &energy_, energy);
}

__global__ void iDMExchangeCUDA_AFM_UpdateField(ManagedMeshCUDA& cuMesh, cuBReal& energy, bool do_reduction)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC<cuReal3>& Heff = *cuMesh.pHeff;

	cuVEC_VC<cuReal3>& M2 = *cuMesh.pM2;
	cuVEC<cuReal3>& Heff2 = *cuMesh.pHeff2;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal energy_ = 0.0;

	if (idx < Heff.linear_size()) {

		cuReal3 Hexch = cuReal3();
		cuReal3 Hexch2 = cuReal3();

		if (M.is_not_empty(idx)) {

			cuReal2 Ms_AFM = *cuMesh.pMs_AFM;
			cuReal2 A_AFM = *cuMesh.pA_AFM;
			cuBReal A12 = *cuMesh.pA12;
			cuReal2 D_AFM = *cuMesh.pD_AFM;
			cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs_AFM, Ms_AFM, *cuMesh.pA_AFM, A_AFM, *cuMesh.pA12, A12, *cuMesh.pD_AFM, D_AFM);

			if (M.is_plane_interior(idx)) {

				//interior point : can use cheaper neu versions

				//direct exchange contribution
				Hexch = 2 * A_AFM.i * M.delsq_neu(idx) / ((cuBReal)MU0 * Ms_AFM.i * Ms_AFM.i) + (4 * A12 / (MU0*Ms_AFM.i*Ms_AFM.j)) * M2[idx];
				Hexch2 = 2 * A_AFM.j * M2.delsq_neu(idx) / ((cuBReal)MU0 * Ms_AFM.j * Ms_AFM.j) + (4 * A12 / (MU0*Ms_AFM.i*Ms_AFM.j)) * M[idx];

				//Dzyaloshinskii-Moriya interfacial exchange contribution

				//Differentials of M components (we only need 4, not all 9 so this could be optimised). First index is the differential direction, second index is the M component
				cuReal33 Mdiff = M.grad_neu(idx);

				//Hdm, ex = -2D / (mu0*Ms) * (dmz / dx, dmz / dy, -dmx / dx - dmy / dy)
				Hexch += -2 * D_AFM.i * cuReal3(Mdiff.x.z, Mdiff.y.z, -Mdiff.x.x - Mdiff.y.y) / ((cuBReal)MU0 * Ms_AFM.i * Ms_AFM.i);

				//same thing on sub-lattice B (2)

				Mdiff = M2.grad_neu(idx);
				Hexch2 += -2 * D_AFM.j * cuReal3(Mdiff.x.z, Mdiff.y.z, -Mdiff.x.x - Mdiff.y.y) / ((cuBReal)MU0 * Ms_AFM.j * Ms_AFM.j);
			}
			else {

				//Non-homogeneous Neumann boundary conditions apply when using DMI. Required to ensure Brown's condition is fulfilled, i.e. m x h -> 0 when relaxing.
				cuReal3 bnd_dm_dx = (D_AFM.i / (2 * A_AFM.i)) * cuReal3(M[idx].z, 0, -M[idx].x);
				cuReal3 bnd_dm_dy = (D_AFM.i / (2 * A_AFM.i)) * cuReal3(0, M[idx].z, -M[idx].y);
				cuReal33 bnd_nneu = cuReal33(bnd_dm_dx, bnd_dm_dy, cuReal3());

				//direct exchange contribution
				Hexch = 2 * A_AFM.i * M.delsq_nneu(idx, bnd_nneu) / ((cuBReal)MU0 * Ms_AFM.i * Ms_AFM.i) + (4 * A12 / (MU0*Ms_AFM.i*Ms_AFM.j)) * M2[idx];

				//Dzyaloshinskii-Moriya interfacial exchange contribution

				//Differentials of M components (we only need 4, not all 9 so this could be optimised). First index is the differential direction, second index is the M component
				cuReal33 Mdiff = M.grad_nneu(idx, bnd_nneu);

				//Hdm, ex = -2D / (mu0*Ms) * (dmz / dx, dmz / dy, -dmx / dx - dmy / dy)
				Hexch += -2 * D_AFM.i * cuReal3(Mdiff.x.z, Mdiff.y.z, -Mdiff.x.x - Mdiff.y.y) / ((cuBReal)MU0 * Ms_AFM.i * Ms_AFM.i);

				//same thing on sub-lattice B (2)

				bnd_dm_dx = (D_AFM.j / (2 * A_AFM.j)) * cuReal3(M2[idx].z, 0, -M2[idx].x);
				bnd_dm_dy = (D_AFM.j / (2 * A_AFM.j)) * cuReal3(0, M2[idx].z, -M2[idx].y);
				bnd_nneu = cuReal33(bnd_dm_dx, bnd_dm_dy, cuReal3());

				Hexch2 = 2 * A_AFM.j * M2.delsq_nneu(idx, bnd_nneu) / ((cuBReal)MU0 * Ms_AFM.j * Ms_AFM.j) + (4 * A12 / (MU0*Ms_AFM.i*Ms_AFM.j)) * M[idx];

				Mdiff = M2.grad_nneu(idx, bnd_nneu);
				Hexch2 += -2 * D_AFM.j * cuReal3(Mdiff.x.z, Mdiff.y.z, -Mdiff.x.x - Mdiff.y.y) / ((cuBReal)MU0 * Ms_AFM.j * Ms_AFM.j);
			}

			if (do_reduction) {

				int non_empty_cells = M.get_nonempty_cells();
				if (non_empty_cells) energy_ = -(cuBReal)MU0 * (M[idx] * Hexch + M2[idx] * Hexch2) / (4 * non_empty_cells);
			}
		}

		Heff[idx] += Hexch;
		Heff2[idx] += Hexch2;
	}

	if (do_reduction) reduction_sum(0, 1, &energy_, energy);
}

//----------------------- UpdateField LAUNCHER

void iDMExchangeCUDA::UpdateField(void)
{
	if (pMeshCUDA->GetMeshType() == MESH_ANTIFERROMAGNETIC) {

		//anti-ferromagnetic mesh

		if (pMeshCUDA->CurrentTimeStepSolved()) {

			ZeroEnergy();

			iDMExchangeCUDA_AFM_UpdateField << < (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >> > (pMeshCUDA->cuMesh, energy, true);
		}
		else {

			iDMExchangeCUDA_AFM_UpdateField << < (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >> > (pMeshCUDA->cuMesh, energy, false);
		}
	}
	else {

		//ferromagnetic mesh

		if (pMeshCUDA->CurrentTimeStepSolved()) {

			ZeroEnergy();

			iDMExchangeCUDA_FM_UpdateField << < (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >> > (pMeshCUDA->cuMesh, energy, true);
		}
		else {

			iDMExchangeCUDA_FM_UpdateField << < (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >> > (pMeshCUDA->cuMesh, energy, false);
		}
	}

	if (pMeshCUDA->GetMeshExchangeCoupling()) CalculateExchangeCoupling(energy);
}

#endif

#endif