#include "hip/hip_runtime.h"
#include "SurfExchangeCUDA.h"

#if COMPILECUDA == 1

#ifdef MODULE_SURFEXCHANGE

#include "BorisCUDALib.cuh"

#include "Mesh_FerromagneticCUDA.h"
#include "MeshParamsControlCUDA.h"

__global__ void SurfExchangeCUDA_Top_UpdateField(ManagedMeshCUDA& cuMesh, ManagedMeshCUDA* pMesh_Top, size_t coupled_meshes, cuBReal& energy, int& coupled_cells, bool do_reduction)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC<cuReal3>& Heff = *cuMesh.pHeff;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuSZ3 n = M.n;
	cuReal3 h = M.h;

	//thickness of layer - SurfExchange applies for layers in the xy plane
	cuBReal thickness = M.rect.e.z - M.rect.s.z;

	cuBReal energy_ = 0.0;

	if (idx < n.x * n.y) {

		int i = idx % n.x;
		int j = idx / n.x;
		int cell_idx = i + j * n.x + (n.z - 1) * n.x*n.y;

		//skip empty cells
		if (M.is_not_empty(cell_idx)) {

			cuBReal Ms = *cuMesh.pMs;
			cuMesh.update_parameters_mcoarse(cell_idx, *cuMesh.pMs, Ms);

			//check all meshes for coupling
			for (int mesh_idx = 0; mesh_idx < coupled_meshes; mesh_idx++) {

				//relative coordinates to read value from top mesh (the one we're coupling to here)
				cuReal3 cell_rel_pos = cuReal3((i + 0.5) * h.x, (j + 0.5) * h.y, pMesh_Top[mesh_idx].pM->h.z / 2);

				cuVEC_VC<cuReal3>& M_Top = *(pMesh_Top[mesh_idx].pM);

				//coupling layer thickness
				cuBReal thickness_top = M_Top.rect.e.z - M_Top.rect.s.z;

				//effective thickness for the coupling equation
				cuBReal thickness_eff = 2 * thickness * thickness_top / (thickness + thickness_top);

				//can't couple to an empty cell
				if (cuIsZ(M_Top[cell_rel_pos].norm())) continue;

				cuBReal J1 = *(pMesh_Top[mesh_idx].pJ1);
				cuBReal J2 = *(pMesh_Top[mesh_idx].pJ2);
				pMesh_Top[mesh_idx].update_parameters_atposition(cell_rel_pos, *(pMesh_Top[mesh_idx].pJ1), J1, *(pMesh_Top[mesh_idx].pJ2), J2);

				//get magnetization value in top mesh cell to couple with
				cuReal3 m_j = M_Top[cell_rel_pos].normalized();
				cuReal3 m_i = M[cell_idx] / Ms;

				cuBReal dot_prod = m_i * m_j;

				//total surface exchange field in coupling cells, including bilinear and biquadratic terms
				cuReal3 Hsurfexh = (m_j / ((cuBReal)MU0 * Ms * thickness_eff)) * (J1 + 2 * J2 * dot_prod);

				Heff[cell_idx] += Hsurfexh;

				if (do_reduction) {

					energy_ = (-1 * J1 - 2 * J2 * dot_prod) * dot_prod / (thickness_eff * coupled_cells);
				}
			}
		}
	}

	if (do_reduction) reduction_sum(0, 1, &energy_, energy);
}

__global__ void SurfExchangeCUDA_Bot_UpdateField(ManagedMeshCUDA& cuMesh, ManagedMeshCUDA* pMesh_Bot, size_t coupled_meshes, cuBReal& energy, int& coupled_cells, bool do_reduction)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC<cuReal3>& Heff = *cuMesh.pHeff;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuSZ3 n = M.n;
	cuReal3 h = M.h;

	//thickness of layer - SurfExchange applies for layers in the xy plane
	cuBReal thickness = M.rect.e.z - M.rect.s.z;

	cuBReal energy_ = 0.0;

	if (idx < n.x * n.y) {

		int i = idx % n.x;
		int j = idx / n.x;
		int cell_idx = i + j * n.x;

		//skip empty cells
		if (M.is_not_empty(cell_idx)) {

			cuBReal Ms = *cuMesh.pMs;
			cuBReal J1 = *cuMesh.pJ1;
			cuBReal J2 = *cuMesh.pJ2;
			cuMesh.update_parameters_mcoarse(cell_idx, *cuMesh.pMs, Ms, *cuMesh.pJ1, J1, *cuMesh.pJ2, J2);

			//check all meshes for coupling
			for (int mesh_idx = 0; mesh_idx < coupled_meshes; mesh_idx++) {

				//relative coordinates to read value from bottom mesh (the one we're coupling to here)
				cuReal3 cell_rel_pos = cuReal3((i + 0.5) * h.x, (j + 0.5) * h.y, pMesh_Bot[mesh_idx].pM->rect.e.z - (pMesh_Bot[mesh_idx].pM->h.z / 2));

				cuVEC_VC<cuReal3>& M_Bot = *(pMesh_Bot[mesh_idx].pM);

				//coupling layer thickness
				cuBReal thickness_bot = M_Bot.rect.e.z - M_Bot.rect.s.z;

				//effective thickness for the coupling equation
				cuBReal thickness_eff = 2 * thickness * thickness_bot / (thickness + thickness_bot);

				//can't couple to an empty cell
				if (cuIsZ(M_Bot[cell_rel_pos].norm())) continue;

				//yes, then get value of magnetization used in coupling with current cell at cell_idx
				cuReal3 m_j = M_Bot[cell_rel_pos].normalized();
				cuReal3 m_i = M[cell_idx] / Ms;

				cuBReal dot_prod = m_i * m_j;

				//total surface exchange field in coupling cells, including bilinear and biquadratic terms
				cuReal3 Hsurfexh = (m_j / ((cuBReal)MU0 * Ms * thickness_eff)) * (J1 + 2 * J2 * dot_prod);

				Heff[cell_idx] += Hsurfexh;

				if (do_reduction) {

					energy_ = (-1 * J1 - 2 * J2 * dot_prod) * dot_prod / (thickness_eff * coupled_cells);
				}
			}
		}
	}

	if (do_reduction) reduction_sum(0, 1, &energy_, energy);
}

//----------------------- UpdateField LAUNCHER

void SurfExchangeCUDA::UpdateField(void)
{
	if (pMeshCUDA->CurrentTimeStepSolved()) {

		ZeroEnergy();

		//Top
		if (pMesh_Top.size()) {

			SurfExchangeCUDA_Top_UpdateField << < (pMeshCUDA->n.x*pMeshCUDA->n.y + CUDATHREADS) / CUDATHREADS, CUDATHREADS >> >
				(pMeshCUDA->cuMesh, pMesh_Top, pMesh_Top.size(), energy, coupled_cells, true);
		}

		//Bottom
		if (pMesh_Bot.size()) {

			SurfExchangeCUDA_Bot_UpdateField << < (pMeshCUDA->n.x*pMeshCUDA->n.y + CUDATHREADS) / CUDATHREADS, CUDATHREADS >> >
				(pMeshCUDA->cuMesh, pMesh_Bot, pMesh_Bot.size(), energy, coupled_cells, true);
		}
	}
	else {

		//Top
		if (pMesh_Top.size()) {

			SurfExchangeCUDA_Top_UpdateField << < (pMeshCUDA->n.x*pMeshCUDA->n.y + CUDATHREADS) / CUDATHREADS, CUDATHREADS >> >
				(pMeshCUDA->cuMesh, pMesh_Top, pMesh_Top.size(), energy, coupled_cells, false);
		}

		//Bottom
		if (pMesh_Bot.size()) {

			SurfExchangeCUDA_Bot_UpdateField << < (pMeshCUDA->n.x*pMeshCUDA->n.y + CUDATHREADS) / CUDATHREADS, CUDATHREADS >> >
				(pMeshCUDA->cuMesh, pMesh_Bot, pMesh_Bot.size(), energy, coupled_cells, false);
		}
	}
}

#endif

#endif

