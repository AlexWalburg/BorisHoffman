#include "hip/hip_runtime.h"
#include "TransportCUDA.h"

#if COMPILECUDA == 1

#ifdef MODULE_COMPILATION_TRANSPORT

#include "BorisCUDALib.cuh"

#include "MeshCUDA.h"
#include "SuperMeshCUDA.h"
#include "MeshParamsControlCUDA.h"

//-------------------Display Calculation Methods

//SPIN CURRENT

__global__ void GetSpinCurrent_Kernel(int component, cuVEC<cuReal3>& displayVEC, ManagedMeshCUDA& cuMesh, TransportCUDA_Spin_S_Funcs& poisson_Spin_S)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC_VC<cuReal3>& S = *cuMesh.pS;
	cuVEC_VC<cuReal3>& E = *cuMesh.pE;
	cuVEC_VC<cuBReal>& elC = *cuMesh.pelC;

	cuVEC_VC<cuReal3>& dM_dt = *poisson_Spin_S.pdM_dt;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < S.linear_size()) {

		bool cpump_enabled = cuIsNZ(cuMesh.pcpump_eff->get0());
		bool the_enabled = cuIsNZ(cuMesh.pthe_eff->get0());

		cuReal33 Js = cuReal33();

		if (S.is_not_empty(idx)) {
			
			if (poisson_Spin_S.stsolve == STSOLVE_FERROMAGNETIC) {

				//magnetic mesh terms

				cuBReal Ms = *cuMesh.pMs;
				cuBReal P = *cuMesh.pP;
				cuBReal De = *cuMesh.pDe;
				cuMesh.update_parameters_ecoarse(idx, *cuMesh.pMs, Ms, *cuMesh.pP, P, *cuMesh.pDe, De);

				//1. drift
				int idx_M = M.position_to_cellidx(S.cellidx_to_position(idx));

				cuReal3 Mval = M[idx_M];
				cuReal33 grad_S = S.grad_neu(idx);

				Js = (E[idx] | Mval) * (P * elC[idx] / Ms) * (-(cuBReal)MUB_E);

				//2. diffusion with homogeneous Neumann boundary condition
				Js -= grad_S * De;

				//3. charge pumping
				//4. topological Hall effect

				if (component != 2 && (cpump_enabled || the_enabled)) {

					cuReal33 grad_m = M.grad_neu(idx_M) / Ms;

					//topological Hall effect contribution
					if (the_enabled) {

						cuBReal n_density = *cuMesh.pn_density;
						cuMesh.update_parameters_ecoarse(idx, *cuMesh.pn_density, n_density);

						cuReal3 B = (grad_m.x ^ grad_m.y);
						Js += cuMesh.pthe_eff->get0() * ((cuBReal)HBAR_E * (cuBReal)MUB_E * elC[idx] * elC[idx] / ((cuBReal)ECHARGE * n_density)) * cuReal33(-E[idx].y * B, E[idx].x * B, cuReal3());
					}

					//charge pumping contribution
					if (cpump_enabled) {

						//value a1
						cuReal3 dm_dt = dM_dt[idx_M] / Ms;
						Js += cuMesh.pcpump_eff->get0() * ((cuBReal)HBAR_E * (cuBReal)MUB_E * elC[idx] / 2) * cuReal33(dm_dt ^ grad_m.x, dm_dt ^ grad_m.y, cuReal3());
					}
				}
			}
			else {

				//non-magnetic mesh terms

				cuBReal De = *cuMesh.pDe;
				cuBReal SHA = *cuMesh.pSHA;
				cuMesh.update_parameters_ecoarse(idx, *cuMesh.pDe, De, *cuMesh.pSHA, SHA);

				//1. SHE contribution
				Js = cu_epsilon3(E[idx]) * SHA * elC[idx] * (cuBReal)MUB_E;

				//2. diffusion with non-homogeneous Neumann boundary condition
				Js -= S.grad_nneu(idx, cu_epsilon3(E[idx]) * (SHA * elC[idx] * (cuBReal)MUB_E / De)) * De;
			}
		}

		switch (component) {

		case 0:
			displayVEC[idx] = Js.x;
			break;
		case 1:
			displayVEC[idx] = Js.y;
			break;
		case 2:
			displayVEC[idx] = Js.z;
			break;
		}
	}
}

//SPIN TORQUE

__global__ void GetSpinTorque_Kernel(cuVEC<cuReal3>& displayVEC, ManagedMeshCUDA& cuMesh)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC_VC<cuReal3>& S = *cuMesh.pS;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < M.linear_size()) {

		if (M.is_empty(idx)) {

			displayVEC[idx] = cuReal3();
			return;
		}

		cuBReal De = *cuMesh.pDe;
		cuBReal ts_eff = *cuMesh.pts_eff;
		cuBReal Ms = *cuMesh.pMs;
		cuBReal l_ex = *cuMesh.pl_ex;
		cuBReal l_ph = *cuMesh.pl_ph;
		cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs, Ms, *cuMesh.pDe, De, *cuMesh.pts_eff, ts_eff, *cuMesh.pl_ex, l_ex, *cuMesh.pl_ph, l_ph);

		cuReal3 Sav = S.weighted_average(M.cellidx_to_position(idx), M.h);

		displayVEC[idx] = ts_eff * ((Sav ^ M[idx]) * De / (Ms * l_ex * l_ex) + (M[idx] ^ (Sav ^ M[idx])) * De / (Ms * Ms * l_ph * l_ph));
	}
}

//SPIN INTERFACE TORQUE

__global__ void CalculateDisplaySAInterfaceTorque_Kernel(CMBNDInfoCUDA& contact, TransportCUDA_Spin_S_Funcs& cmbndFuncs_sec, TransportCUDA_Spin_S_Funcs& cmbndFuncs_pri, cuVEC<cuReal3>& displayVEC)
{
	cuVEC_VC<cuReal3>& M = *cmbndFuncs_pri.pcuMesh->pM;
	cuVEC_VC<cuReal3>& S_pri = *cmbndFuncs_pri.pcuMesh->pS;
	cuVEC_VC<cuReal3>& S_sec = *cmbndFuncs_sec.pcuMesh->pS;

	int box_idx = blockIdx.x * blockDim.x + threadIdx.x;

	//interface conductance method with F being the primary mesh : calculate and set spin torque

	//convert the cells box from S mesh to M mesh
	cuINT3 mbox_start = M.cellidx_from_position(S_pri.cellidx_to_position(contact.cells_box.s) + M.rect.s);
	cuINT3 mbox_end = M.cellidx_from_position(S_pri.cellidx_to_position(contact.cells_box.e - cuINT3(1)) + M.rect.s) + cuINT3(1);

	if ((mbox_end.i - mbox_start.i) == 0) mbox_end.i = mbox_start.i + 1;
	if ((mbox_end.j - mbox_start.j) == 0) mbox_end.j = mbox_start.j + 1;
	if ((mbox_end.k - mbox_start.k) == 0) mbox_end.k = mbox_start.k + 1;

	cuINT3 box_sizes = mbox_end - mbox_start;

	if (box_idx < box_sizes.dim()) {

		//the cellsize perpendicular to the contact (in the M mesh)
		cuBReal dh = (cuReal3(contact.cell_shift) & M.h).norm();

		int i = (box_idx % box_sizes.x) + mbox_start.i;
		int j = ((box_idx / box_sizes.x) % box_sizes.y) + mbox_start.j;
		int k = (box_idx / (box_sizes.x * box_sizes.y)) + mbox_start.k;

		//index of magnetic cell 1
		int mcell1_idx = i + j * M.n.x + k * M.n.x*M.n.y;

		if (M.is_empty(mcell1_idx)) return;

		cuBReal Ms = *cmbndFuncs_pri.pcuMesh->pMs;
		cuBReal tsi_eff = *cmbndFuncs_pri.pcuMesh->ptsi_eff;
		cmbndFuncs_pri.pcuMesh->update_parameters_mcoarse(mcell1_idx, *cmbndFuncs_pri.pcuMesh->pMs, Ms, *cmbndFuncs_pri.pcuMesh->ptsi_eff, tsi_eff);

		//position at interface relative to primary mesh
		cuReal3 mhshift_primary = contact.hshift_primary.normalized() & M.h;
		cuReal3 relpos_interf = ((cuReal3(i, j, k) + cuReal3(0.5)) & M.h) + mhshift_primary / 2;

		cuReal3 relpos_1 = relpos_interf - contact.hshift_primary / 2;

		cuReal3 relpos_m1 = S_pri.rect.s - S_sec.rect.s + relpos_interf + contact.hshift_secondary / 2;

		cuReal3 stencil = M.h - cu_mod(mhshift_primary) + cu_mod(contact.hshift_secondary);

		//S values
		cuReal3 S_1 = S_pri.weighted_average(relpos_1, stencil);
		cuReal3 S_2 = S_pri.weighted_average(relpos_1 - contact.hshift_primary, stencil);
		cuReal3 S_m1 = S_sec.weighted_average(relpos_m1, stencil);
		cuReal3 S_m2 = S_sec.weighted_average(relpos_m1 + contact.hshift_secondary, stencil);

		//c values
		cuBReal c_m1 = cmbndFuncs_sec.c_func_sec(relpos_m1, stencil);
		cuBReal c_m2 = cmbndFuncs_sec.c_func_sec(relpos_m1 + contact.hshift_secondary, stencil);
		cuBReal c_1 = cmbndFuncs_pri.c_func_sec(relpos_1, stencil);
		cuBReal c_2 = cmbndFuncs_pri.c_func_sec(relpos_1 - contact.hshift_primary, stencil);

		//Calculate S drop at the interface
		cuReal3 Vs_F = 1.5 * c_1 * S_1 - 0.5 * c_2 * S_2;
		cuReal3 Vs_N = 1.5 * c_m1 * S_m1 - 0.5 * c_m2 * S_m2;
		cuReal3 dVs = Vs_F - Vs_N;

		//Get G values from top contacting mesh
		cuReal2 Gmix;
		if (contact.IsPrimaryTop()) {

			Gmix = *cmbndFuncs_pri.pcuMesh->pGmix;
			cmbndFuncs_pri.pcuMesh->update_parameters_mcoarse(mcell1_idx, *cmbndFuncs_pri.pcuMesh->pGmix, Gmix);
		}
		else {

			Gmix = *cmbndFuncs_sec.pcuMesh->pGmix;
			cmbndFuncs_sec.pcuMesh->update_parameters_atposition(relpos_m1, *cmbndFuncs_sec.pcuMesh->pGmix, Gmix);
		}

		cuBReal gI = (2.0 * (cuBReal)GMUB_2E / dh) * Gmix.j / Ms;
		cuBReal gR = (2.0 * (cuBReal)GMUB_2E / dh) * Gmix.i / Ms;

		displayVEC[mcell1_idx] += tsi_eff * (gI * (M[mcell1_idx] ^ dVs) + gR * (M[mcell1_idx] ^ (M[mcell1_idx] ^ dVs)) / Ms);
	}
}

//Launchers

//prepare displayVEC ready for calculation of display quantity
bool TransportCUDA::PrepareDisplayVEC(DBL3 cellsize)
{
	if (pSMeshCUDA->SolveSpinCurrent() && pMeshCUDA->EComputation_Enabled()) {

		//make sure memory is allocated to the correct size
		displayVEC()->assign(cellsize, pMeshCUDA->meshRect, cuReal3(0.0));

		return true;
	}
	else displayVEC()->clear();

	return false;
}

//return x, y, or z component of spin current (component = 0, 1, or 2)
cu_obj<cuVEC<cuReal3>>& TransportCUDA::GetSpinCurrent(int component)
{
	if (!PrepareDisplayVEC(pMeshCUDA->h_e)) return displayVEC;

	if (stsolve != STSOLVE_NONE) {

		GetSpinCurrent_Kernel <<< (pMeshCUDA->n_e.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> (component, displayVEC, pMeshCUDA->cuMesh, poisson_Spin_S);
	}

	return displayVEC;
}

//return spin torque computed from spin accumulation
cu_obj<cuVEC<cuReal3>>& TransportCUDA::GetSpinTorque(void)
{
	if (!PrepareDisplayVEC(pMeshCUDA->h)) return displayVEC;
	
	if (stsolve == STSOLVE_FERROMAGNETIC) {

		GetSpinTorque_Kernel <<< (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> (displayVEC, pMeshCUDA->cuMesh);
	}

	return displayVEC;
}

//Calculate the interface spin accumulation torque for a given contact (in magnetic meshes for NF interfaces with G interface conductance set), accumulating result in displayVEC
void TransportCUDA::CalculateDisplaySAInterfaceTorque(TransportCUDA* ptrans_sec, CMBNDInfoCUDA& contactCUDA, bool primary_top)
{
	//the top contacting mesh sets G values
	bool isGInterface_Enabled = ((primary_top && pMeshCUDA->GInterface_Enabled()) || (!primary_top && ptrans_sec->pMeshCUDA->GInterface_Enabled()));

	if (stsolve == STSOLVE_FERROMAGNETIC && ptrans_sec->Get_STSolveType() == STSOLVE_NORMALMETAL && isGInterface_Enabled) {

		CalculateDisplaySAInterfaceTorque_Kernel << < (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >> > (contactCUDA, ptrans_sec->poisson_Spin_S, poisson_Spin_S, displayVEC);
	}
}

#endif

#endif