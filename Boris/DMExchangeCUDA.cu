#include "hip/hip_runtime.h"
#include "DMExchangeCUDA.h"

#if COMPILECUDA == 1

#ifdef MODULE_COMPILATION_DMEXCHANGE

#include "BorisCUDALib.cuh"

#include "MeshCUDA.h"
#include "MeshParamsControlCUDA.h"
#include "MeshDefs.h"

//////////////////////////////////////////////////////////////////////// UPDATE FIELD

__global__ void DMExchangeCUDA_FM_UpdateField(ManagedMeshCUDA& cuMesh, cuBReal& energy, bool do_reduction)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC<cuReal3>& Heff = *cuMesh.pHeff;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal energy_ = 0.0;

	if (idx < Heff.linear_size()) {

		cuReal3 Hexch = cuReal3();

		if (M.is_not_empty(idx)) {

			cuBReal Ms = *cuMesh.pMs;
			cuBReal A = *cuMesh.pA;
			cuBReal D = *cuMesh.pD;
			cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs, Ms, *cuMesh.pA, A, *cuMesh.pD, D);

			if (M.is_interior(idx)) {

				//interior point : can use cheaper neu versions

				//direct exchange contribution
				Hexch = 2 * A * M.delsq_neu(idx) / ((cuBReal)MU0 * Ms * Ms);

				//Dzyaloshinskii-Moriya exchange contribution

				//Hdm, ex = -2D / (mu0*Ms) * curl m
				Hexch += -2 * D * M.curl_neu(idx) / ((cuBReal)MU0 * Ms * Ms);
			}
			else {

				//Non-homogeneous Neumann boundary conditions apply when using DMI. Required to ensure Brown's condition is fulfilled, i.e. m x h -> 0 when relaxing.
				cuReal3 bnd_dm_dx = (D / (2 * A)) * cuReal3(0, -M[idx].z, M[idx].y);
				cuReal3 bnd_dm_dy = (D / (2 * A)) * cuReal3(M[idx].z, 0, -M[idx].x);
				cuReal3 bnd_dm_dz = (D / (2 * A)) * cuReal3(-M[idx].y, M[idx].x, 0);
				cuReal33 bnd_nneu = cuReal33(bnd_dm_dx, bnd_dm_dy, bnd_dm_dz);

				//direct exchange contribution
				Hexch = 2 * A * M.delsq_nneu(idx, bnd_nneu) / ((cuBReal)MU0 * Ms * Ms);

				//Dzyaloshinskii-Moriya exchange contribution

				//Hdm, ex = -2D / (mu0*Ms) * curl m
				Hexch += -2 * D * M.curl_nneu(idx, bnd_nneu) / ((cuBReal)MU0 * Ms * Ms);
			}

			if (do_reduction) {

				int non_empty_cells = M.get_nonempty_cells();
				if (non_empty_cells) energy_ = -(cuBReal)MU0 * M[idx] * Hexch / (2 * non_empty_cells);
			}
		}

		Heff[idx] += Hexch;
	}

	if (do_reduction) reduction_sum(0, 1, &energy_, energy);
}

__global__ void DMExchangeCUDA_AFM_UpdateField(ManagedMeshCUDA& cuMesh, cuBReal& energy, bool do_reduction)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC<cuReal3>& Heff = *cuMesh.pHeff;

	cuVEC_VC<cuReal3>& M2 = *cuMesh.pM2;
	cuVEC<cuReal3>& Heff2 = *cuMesh.pHeff2;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal energy_ = 0.0;

	if (idx < Heff.linear_size()) {

		cuReal3 Hexch = cuReal3();
		cuReal3 Hexch2 = cuReal3();

		if (M.is_not_empty(idx)) {

			cuReal2 Ms_AFM = *cuMesh.pMs_AFM;
			cuReal2 A_AFM = *cuMesh.pA_AFM;
			cuReal2 Ah = *cuMesh.pAh;
			cuReal2 Anh = *cuMesh.pAnh;
			cuReal2 D_AFM = *cuMesh.pD_AFM;
			cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs_AFM, Ms_AFM, *cuMesh.pA_AFM, A_AFM, *cuMesh.pAh, Ah, *cuMesh.pAnh, Anh, *cuMesh.pD_AFM, D_AFM);

			if (M.is_interior(idx)) {

				//interior point : can use cheaper neu versions

				//1. direct exchange contribution + AFM contribution
				cuReal3 delsq_M_A = M.delsq_neu(idx);
				cuReal3 delsq_M_B = M2.delsq_neu(idx);

				cuReal2 Mmag = cuReal2(M[idx].norm(), M2[idx].norm());

				Hexch = 2 * A_AFM.i * delsq_M_A / ((cuBReal)MU0 * Ms_AFM.i * Ms_AFM.i) + (-4 * Ah.i * (M[idx] ^ (M[idx] ^ M2[idx])) / (Mmag.i*Mmag.i) + Anh.i * delsq_M_B) / ((cuBReal)MU0*Ms_AFM.i*Ms_AFM.j);
				Hexch2 = 2 * A_AFM.j * delsq_M_B / ((cuBReal)MU0 * Ms_AFM.j * Ms_AFM.j) + (-4 * Ah.j * (M2[idx] ^ (M2[idx] ^ M[idx])) / (Mmag.j*Mmag.j) + Anh.j * delsq_M_A) / ((cuBReal)MU0*Ms_AFM.i*Ms_AFM.j);

				//Dzyaloshinskii-Moriya exchange contribution

				//Hdm, ex = -2D / (mu0*Ms) * curl m
				Hexch += -2 * D_AFM.i * M.curl_neu(idx) / ((cuBReal)MU0 * Ms_AFM.i * Ms_AFM.i);
				Hexch2 += -2 * D_AFM.j * M2.curl_neu(idx) / ((cuBReal)MU0 * Ms_AFM.j * Ms_AFM.j);
			}
			else {

				//Non-homogeneous Neumann boundary conditions apply when using DMI. Required to ensure Brown's condition is fulfilled, i.e. m x h -> 0 when relaxing.
				cuReal3 bnd_dm_dx = (D_AFM.i / (2 * A_AFM.i)) * cuReal3(0, -M[idx].z, M[idx].y);
				cuReal3 bnd_dm_dy = (D_AFM.i / (2 * A_AFM.i)) * cuReal3(M[idx].z, 0, -M[idx].x);
				cuReal3 bnd_dm_dz = (D_AFM.i / (2 * A_AFM.i)) * cuReal3(-M[idx].y, M[idx].x, 0);
				cuReal33 bndA_nneu = cuReal33(bnd_dm_dx, bnd_dm_dy, bnd_dm_dz);

				bnd_dm_dx = (D_AFM.j / (2 * A_AFM.j)) * cuReal3(0, -M2[idx].z, M2[idx].y);
				bnd_dm_dy = (D_AFM.j / (2 * A_AFM.j)) * cuReal3(M2[idx].z, 0, -M2[idx].x);
				bnd_dm_dz = (D_AFM.j / (2 * A_AFM.j)) * cuReal3(-M2[idx].y, M2[idx].x, 0);
				cuReal33 bndB_nneu = cuReal33(bnd_dm_dx, bnd_dm_dy, bnd_dm_dz);

				cuReal3 delsq_M_A = M.delsq_nneu(idx, bndA_nneu);
				cuReal3 delsq_M_B = M2.delsq_nneu(idx, bndB_nneu);

				cuReal2 Mmag = cuReal2(M[idx].norm(), M2[idx].norm());

				//1. direct exchange contribution + AFM contribution

				//cells marked with cmbnd are calculated using exchange coupling to other ferromagnetic meshes - see below; the delsq_nneu evaluates to zero in the CMBND coupling direction.
				Hexch = 2 * A_AFM.i * delsq_M_A / ((cuBReal)MU0 * Ms_AFM.i * Ms_AFM.i) + (-4 * Ah.i * (M[idx] ^ (M[idx] ^ M2[idx])) / (Mmag.i*Mmag.i) + Anh.i * delsq_M_B) / ((cuBReal)MU0*Ms_AFM.i*Ms_AFM.j);
				Hexch2 = 2 * A_AFM.j * delsq_M_B / ((cuBReal)MU0 * Ms_AFM.j * Ms_AFM.j) + (-4 * Ah.j * (M2[idx] ^ (M2[idx] ^ M[idx])) / (Mmag.j*Mmag.j) + Anh.j * delsq_M_A) / ((cuBReal)MU0*Ms_AFM.i*Ms_AFM.j);

				//2. Dzyaloshinskii-Moriya exchange contribution

				//Hdm, ex = -2D / (mu0*Ms) * curl m
				//For cmbnd cells curl_nneu does not evaluate to zero in the CMBND coupling direction, but sided differentials are used - when setting values at CMBND cells for exchange coupled meshes must correct for this.
				Hexch += -2 * D_AFM.i * M.curl_nneu(idx, bndA_nneu) / ((cuBReal)MU0 * Ms_AFM.i * Ms_AFM.i);
				Hexch2 += -2 * D_AFM.j * M2.curl_nneu(idx, bndB_nneu) / ((cuBReal)MU0 * Ms_AFM.j * Ms_AFM.j);
			}

			if (do_reduction) {

				int non_empty_cells = M.get_nonempty_cells();
				if (non_empty_cells) energy_ = -(cuBReal)MU0 * (M[idx] * Hexch + M2[idx] * Hexch2) / (4 * non_empty_cells);
			}
		}

		Heff[idx] += Hexch;
		Heff2[idx] += Hexch2;
	}

	if (do_reduction) reduction_sum(0, 1, &energy_, energy);
}

//----------------------- UpdateField LAUNCHER

void DMExchangeCUDA::UpdateField(void)
{

	if (pMeshCUDA->GetMeshType() == MESH_ANTIFERROMAGNETIC) {

		//anti-ferromagnetic mesh

		if (pMeshCUDA->CurrentTimeStepSolved()) {

			ZeroEnergy();

			DMExchangeCUDA_AFM_UpdateField << < (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >> > (pMeshCUDA->cuMesh, energy, true);
		}
		else {

			DMExchangeCUDA_AFM_UpdateField << < (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >> > (pMeshCUDA->cuMesh, energy, false);
		}
	}
	else {

		//ferromagnetic mesh

		if (pMeshCUDA->CurrentTimeStepSolved()) {

			ZeroEnergy();

			DMExchangeCUDA_FM_UpdateField << < (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >> > (pMeshCUDA->cuMesh, energy, true);
		}
		else {

			DMExchangeCUDA_FM_UpdateField << < (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >> > (pMeshCUDA->cuMesh, energy, false);
		}
	}

	if (pMeshCUDA->GetMeshExchangeCoupling()) CalculateExchangeCoupling(energy);
}

//////////////////////////////////////////////////////////////////////// ENERGY DENSITY DATA METHODS

__global__ void DMExchangeCUDA_FM_GetEnergy(ManagedMeshCUDA& cuMesh, cuBReal& energy, size_t& points_count, cuRect avRect)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal energy_ = 0.0;

	bool include_in_reduction = false;

	if (idx < M.linear_size()) {

		cuReal3 Hexch = cuReal3();

		if (M.is_not_empty(idx) && avRect.contains(M.cellidx_to_position(idx))) {

			cuBReal Ms = *cuMesh.pMs;
			cuBReal A = *cuMesh.pA;
			cuBReal D = *cuMesh.pD;
			cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs, Ms, *cuMesh.pA, A, *cuMesh.pD, D);

			if (M.is_interior(idx)) {

				//interior point : can use cheaper neu versions

				//direct exchange contribution
				Hexch = 2 * A * M.delsq_neu(idx) / ((cuBReal)MU0 * Ms * Ms);

				//Dzyaloshinskii-Moriya exchange contribution

				//Hdm, ex = -2D / (mu0*Ms) * curl m
				Hexch += -2 * D * M.curl_neu(idx) / ((cuBReal)MU0 * Ms * Ms);
			}
			else {

				//Non-homogeneous Neumann boundary conditions apply when using DMI. Required to ensure Brown's condition is fulfilled, i.e. m x h -> 0 when relaxing.
				cuReal3 bnd_dm_dx = (D / (2 * A)) * cuReal3(0, -M[idx].z, M[idx].y);
				cuReal3 bnd_dm_dy = (D / (2 * A)) * cuReal3(M[idx].z, 0, -M[idx].x);
				cuReal3 bnd_dm_dz = (D / (2 * A)) * cuReal3(-M[idx].y, M[idx].x, 0);
				cuReal33 bnd_nneu = cuReal33(bnd_dm_dx, bnd_dm_dy, bnd_dm_dz);

				//direct exchange contribution
				Hexch = 2 * A * M.delsq_nneu(idx, bnd_nneu) / ((cuBReal)MU0 * Ms * Ms);

				//Dzyaloshinskii-Moriya exchange contribution

				//Hdm, ex = -2D / (mu0*Ms) * curl m
				Hexch += -2 * D * M.curl_nneu(idx, bnd_nneu) / ((cuBReal)MU0 * Ms * Ms);
			}

			energy_ = -(cuBReal)MU0 * M[idx] * Hexch / 2;
			include_in_reduction = true;
		}
	}

	reduction_avg(0, 1, &energy_, energy, points_count, include_in_reduction);
}

__global__ void DMExchangeCUDA_AFM_GetEnergy(ManagedMeshCUDA& cuMesh, cuBReal& energy, size_t& points_count, cuRect avRect)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC_VC<cuReal3>& M2 = *cuMesh.pM2;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal energy_ = 0.0;

	bool include_in_reduction = false;

	if (idx < M.linear_size()) {

		cuReal3 Hexch = cuReal3();
		cuReal3 Hexch2 = cuReal3();

		if (M.is_not_empty(idx) && avRect.contains(M.cellidx_to_position(idx))) {

			cuReal2 Ms_AFM = *cuMesh.pMs_AFM;
			cuReal2 A_AFM = *cuMesh.pA_AFM;
			cuReal2 Ah = *cuMesh.pAh;
			cuReal2 Anh = *cuMesh.pAnh;
			cuReal2 D_AFM = *cuMesh.pD_AFM;
			cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs_AFM, Ms_AFM, *cuMesh.pA_AFM, A_AFM, *cuMesh.pAh, Ah, *cuMesh.pAnh, Anh, *cuMesh.pD_AFM, D_AFM);

			if (M.is_interior(idx)) {

				//interior point : can use cheaper neu versions

				//1. direct exchange contribution + AFM contribution
				cuReal3 delsq_M_A = M.delsq_neu(idx);
				cuReal3 delsq_M_B = M2.delsq_neu(idx);

				cuReal2 Mmag = cuReal2(M[idx].norm(), M2[idx].norm());

				Hexch = 2 * A_AFM.i * delsq_M_A / ((cuBReal)MU0 * Ms_AFM.i * Ms_AFM.i) + (-4 * Ah.i * (M[idx] ^ (M[idx] ^ M2[idx])) / (Mmag.i*Mmag.i) + Anh.i * delsq_M_B) / ((cuBReal)MU0*Ms_AFM.i*Ms_AFM.j);
				Hexch2 = 2 * A_AFM.j * delsq_M_B / ((cuBReal)MU0 * Ms_AFM.j * Ms_AFM.j) + (-4 * Ah.j * (M2[idx] ^ (M2[idx] ^ M[idx])) / (Mmag.j*Mmag.j) + Anh.j * delsq_M_A) / ((cuBReal)MU0*Ms_AFM.i*Ms_AFM.j);

				//Dzyaloshinskii-Moriya exchange contribution

				//Hdm, ex = -2D / (mu0*Ms) * curl m
				Hexch += -2 * D_AFM.i * M.curl_neu(idx) / ((cuBReal)MU0 * Ms_AFM.i * Ms_AFM.i);
				Hexch2 += -2 * D_AFM.j * M2.curl_neu(idx) / ((cuBReal)MU0 * Ms_AFM.j * Ms_AFM.j);
			}
			else {

				//Non-homogeneous Neumann boundary conditions apply when using DMI. Required to ensure Brown's condition is fulfilled, i.e. m x h -> 0 when relaxing.
				cuReal3 bnd_dm_dx = (D_AFM.i / (2 * A_AFM.i)) * cuReal3(0, -M[idx].z, M[idx].y);
				cuReal3 bnd_dm_dy = (D_AFM.i / (2 * A_AFM.i)) * cuReal3(M[idx].z, 0, -M[idx].x);
				cuReal3 bnd_dm_dz = (D_AFM.i / (2 * A_AFM.i)) * cuReal3(-M[idx].y, M[idx].x, 0);
				cuReal33 bndA_nneu = cuReal33(bnd_dm_dx, bnd_dm_dy, bnd_dm_dz);

				bnd_dm_dx = (D_AFM.j / (2 * A_AFM.j)) * cuReal3(0, -M2[idx].z, M2[idx].y);
				bnd_dm_dy = (D_AFM.j / (2 * A_AFM.j)) * cuReal3(M2[idx].z, 0, -M2[idx].x);
				bnd_dm_dz = (D_AFM.j / (2 * A_AFM.j)) * cuReal3(-M2[idx].y, M2[idx].x, 0);
				cuReal33 bndB_nneu = cuReal33(bnd_dm_dx, bnd_dm_dy, bnd_dm_dz);

				cuReal3 delsq_M_A = M.delsq_nneu(idx, bndA_nneu);
				cuReal3 delsq_M_B = M2.delsq_nneu(idx, bndB_nneu);

				cuReal2 Mmag = cuReal2(M[idx].norm(), M2[idx].norm());

				//1. direct exchange contribution + AFM contribution

				//cells marked with cmbnd are calculated using exchange coupling to other ferromagnetic meshes - see below; the delsq_nneu evaluates to zero in the CMBND coupling direction.
				Hexch = 2 * A_AFM.i * delsq_M_A / ((cuBReal)MU0 * Ms_AFM.i * Ms_AFM.i) + (-4 * Ah.i * (M[idx] ^ (M[idx] ^ M2[idx])) / (Mmag.i*Mmag.i) + Anh.i * delsq_M_B) / ((cuBReal)MU0*Ms_AFM.i*Ms_AFM.j);
				Hexch2 = 2 * A_AFM.j * delsq_M_B / ((cuBReal)MU0 * Ms_AFM.j * Ms_AFM.j) + (-4 * Ah.j * (M2[idx] ^ (M2[idx] ^ M[idx])) / (Mmag.j*Mmag.j) + Anh.j * delsq_M_A) / ((cuBReal)MU0*Ms_AFM.i*Ms_AFM.j);

				//2. Dzyaloshinskii-Moriya exchange contribution

				//Hdm, ex = -2D / (mu0*Ms) * curl m
				//For cmbnd cells curl_nneu does not evaluate to zero in the CMBND coupling direction, but sided differentials are used - when setting values at CMBND cells for exchange coupled meshes must correct for this.
				Hexch += -2 * D_AFM.i * M.curl_nneu(idx, bndA_nneu) / ((cuBReal)MU0 * Ms_AFM.i * Ms_AFM.i);
				Hexch2 += -2 * D_AFM.j * M2.curl_nneu(idx, bndB_nneu) / ((cuBReal)MU0 * Ms_AFM.j * Ms_AFM.j);
			}

			energy_ = -(cuBReal)MU0 * (M[idx] * Hexch + M2[idx] * Hexch2) / 4;
			include_in_reduction = true;
		}
	}

	reduction_avg(0, 1, &energy_, energy, points_count, include_in_reduction);
}

__global__ void DMExchangeCUDA_FM_GetEnergy_Max(ManagedMeshCUDA& cuMesh, cuBReal& energy, cuRect rectangle)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal energy_ = 0.0;

	bool include_in_reduction = false;

	if (idx < M.linear_size()) {

		cuReal3 Hexch = cuReal3();

		if (M.is_not_empty(idx) && rectangle.contains(M.cellidx_to_position(idx))) {

			cuBReal Ms = *cuMesh.pMs;
			cuBReal A = *cuMesh.pA;
			cuBReal D = *cuMesh.pD;
			cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs, Ms, *cuMesh.pA, A, *cuMesh.pD, D);

			if (M.is_interior(idx)) {

				//interior point : can use cheaper neu versions

				//direct exchange contribution
				Hexch = 2 * A * M.delsq_neu(idx) / ((cuBReal)MU0 * Ms * Ms);

				//Dzyaloshinskii-Moriya exchange contribution

				//Hdm, ex = -2D / (mu0*Ms) * curl m
				Hexch += -2 * D * M.curl_neu(idx) / ((cuBReal)MU0 * Ms * Ms);
			}
			else {

				//Non-homogeneous Neumann boundary conditions apply when using DMI. Required to ensure Brown's condition is fulfilled, i.e. m x h -> 0 when relaxing.
				cuReal3 bnd_dm_dx = (D / (2 * A)) * cuReal3(0, -M[idx].z, M[idx].y);
				cuReal3 bnd_dm_dy = (D / (2 * A)) * cuReal3(M[idx].z, 0, -M[idx].x);
				cuReal3 bnd_dm_dz = (D / (2 * A)) * cuReal3(-M[idx].y, M[idx].x, 0);
				cuReal33 bnd_nneu = cuReal33(bnd_dm_dx, bnd_dm_dy, bnd_dm_dz);

				//direct exchange contribution
				Hexch = 2 * A * M.delsq_nneu(idx, bnd_nneu) / ((cuBReal)MU0 * Ms * Ms);

				//Dzyaloshinskii-Moriya exchange contribution

				//Hdm, ex = -2D / (mu0*Ms) * curl m
				Hexch += -2 * D * M.curl_nneu(idx, bnd_nneu) / ((cuBReal)MU0 * Ms * Ms);
			}

			energy_ = fabs((cuBReal)MU0 * M[idx] * Hexch / 2);
			include_in_reduction = true;
		}
	}

	reduction_max(0, 1, &energy_, energy, include_in_reduction);
}

__global__ void DMExchangeCUDA_AFM_GetEnergy_Max(ManagedMeshCUDA& cuMesh, cuBReal& energy, cuRect rectangle)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC_VC<cuReal3>& M2 = *cuMesh.pM2;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal energy_ = 0.0;

	bool include_in_reduction = false;

	if (idx < M.linear_size()) {

		cuReal3 Hexch = cuReal3();
		cuReal3 Hexch2 = cuReal3();

		if (M.is_not_empty(idx) && rectangle.contains(M.cellidx_to_position(idx))) {

			cuReal2 Ms_AFM = *cuMesh.pMs_AFM;
			cuReal2 A_AFM = *cuMesh.pA_AFM;
			cuReal2 Ah = *cuMesh.pAh;
			cuReal2 Anh = *cuMesh.pAnh;
			cuReal2 D_AFM = *cuMesh.pD_AFM;
			cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs_AFM, Ms_AFM, *cuMesh.pA_AFM, A_AFM, *cuMesh.pAh, Ah, *cuMesh.pAnh, Anh, *cuMesh.pD_AFM, D_AFM);

			if (M.is_interior(idx)) {

				//interior point : can use cheaper neu versions

				//1. direct exchange contribution + AFM contribution
				cuReal3 delsq_M_A = M.delsq_neu(idx);
				cuReal3 delsq_M_B = M2.delsq_neu(idx);

				cuReal2 Mmag = cuReal2(M[idx].norm(), M2[idx].norm());

				Hexch = 2 * A_AFM.i * delsq_M_A / ((cuBReal)MU0 * Ms_AFM.i * Ms_AFM.i) + (-4 * Ah.i * (M[idx] ^ (M[idx] ^ M2[idx])) / (Mmag.i*Mmag.i) + Anh.i * delsq_M_B) / ((cuBReal)MU0*Ms_AFM.i*Ms_AFM.j);
				Hexch2 = 2 * A_AFM.j * delsq_M_B / ((cuBReal)MU0 * Ms_AFM.j * Ms_AFM.j) + (-4 * Ah.j * (M2[idx] ^ (M2[idx] ^ M[idx])) / (Mmag.j*Mmag.j) + Anh.j * delsq_M_A) / ((cuBReal)MU0*Ms_AFM.i*Ms_AFM.j);

				//Dzyaloshinskii-Moriya exchange contribution

				//Hdm, ex = -2D / (mu0*Ms) * curl m
				Hexch += -2 * D_AFM.i * M.curl_neu(idx) / ((cuBReal)MU0 * Ms_AFM.i * Ms_AFM.i);
				Hexch2 += -2 * D_AFM.j * M2.curl_neu(idx) / ((cuBReal)MU0 * Ms_AFM.j * Ms_AFM.j);
			}
			else {

				//Non-homogeneous Neumann boundary conditions apply when using DMI. Required to ensure Brown's condition is fulfilled, i.e. m x h -> 0 when relaxing.
				cuReal3 bnd_dm_dx = (D_AFM.i / (2 * A_AFM.i)) * cuReal3(0, -M[idx].z, M[idx].y);
				cuReal3 bnd_dm_dy = (D_AFM.i / (2 * A_AFM.i)) * cuReal3(M[idx].z, 0, -M[idx].x);
				cuReal3 bnd_dm_dz = (D_AFM.i / (2 * A_AFM.i)) * cuReal3(-M[idx].y, M[idx].x, 0);
				cuReal33 bndA_nneu = cuReal33(bnd_dm_dx, bnd_dm_dy, bnd_dm_dz);

				bnd_dm_dx = (D_AFM.j / (2 * A_AFM.j)) * cuReal3(0, -M2[idx].z, M2[idx].y);
				bnd_dm_dy = (D_AFM.j / (2 * A_AFM.j)) * cuReal3(M2[idx].z, 0, -M2[idx].x);
				bnd_dm_dz = (D_AFM.j / (2 * A_AFM.j)) * cuReal3(-M2[idx].y, M2[idx].x, 0);
				cuReal33 bndB_nneu = cuReal33(bnd_dm_dx, bnd_dm_dy, bnd_dm_dz);

				cuReal3 delsq_M_A = M.delsq_nneu(idx, bndA_nneu);
				cuReal3 delsq_M_B = M2.delsq_nneu(idx, bndB_nneu);

				cuReal2 Mmag = cuReal2(M[idx].norm(), M2[idx].norm());

				//1. direct exchange contribution + AFM contribution

				//cells marked with cmbnd are calculated using exchange coupling to other ferromagnetic meshes - see below; the delsq_nneu evaluates to zero in the CMBND coupling direction.
				Hexch = 2 * A_AFM.i * delsq_M_A / ((cuBReal)MU0 * Ms_AFM.i * Ms_AFM.i) + (-4 * Ah.i * (M[idx] ^ (M[idx] ^ M2[idx])) / (Mmag.i*Mmag.i) + Anh.i * delsq_M_B) / ((cuBReal)MU0*Ms_AFM.i*Ms_AFM.j);
				Hexch2 = 2 * A_AFM.j * delsq_M_B / ((cuBReal)MU0 * Ms_AFM.j * Ms_AFM.j) + (-4 * Ah.j * (M2[idx] ^ (M2[idx] ^ M[idx])) / (Mmag.j*Mmag.j) + Anh.j * delsq_M_A) / ((cuBReal)MU0*Ms_AFM.i*Ms_AFM.j);

				//2. Dzyaloshinskii-Moriya exchange contribution

				//Hdm, ex = -2D / (mu0*Ms) * curl m
				//For cmbnd cells curl_nneu does not evaluate to zero in the CMBND coupling direction, but sided differentials are used - when setting values at CMBND cells for exchange coupled meshes must correct for this.
				Hexch += -2 * D_AFM.i * M.curl_nneu(idx, bndA_nneu) / ((cuBReal)MU0 * Ms_AFM.i * Ms_AFM.i);
				Hexch2 += -2 * D_AFM.j * M2.curl_nneu(idx, bndB_nneu) / ((cuBReal)MU0 * Ms_AFM.j * Ms_AFM.j);
			}

			energy_ = fabs((cuBReal)MU0 * (M[idx] * Hexch + M2[idx] * Hexch2) / 4);
			include_in_reduction = true;
		}
	}

	reduction_max(0, 1, &energy_, energy, include_in_reduction);
}

cuBReal DMExchangeCUDA::GetEnergyDensity(cuRect avRect)
{
	ZeroEnergy();

	if (pMeshCUDA->GetMeshType() == MESH_ANTIFERROMAGNETIC) {

		//anti-ferromagnetic mesh

		DMExchangeCUDA_AFM_GetEnergy <<< (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> (pMeshCUDA->cuMesh, energy, points_count, avRect);
	}
	else {

		//ferromagnetic mesh

		DMExchangeCUDA_FM_GetEnergy <<< (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> (pMeshCUDA->cuMesh, energy, points_count, avRect);
	}

	size_t points_count_cpu = points_count.to_cpu();

	if (points_count_cpu) return energy.to_cpu() / points_count_cpu;
	else return 0.0;
}

cuBReal DMExchangeCUDA::GetEnergy_Max(cuRect rectangle)
{
	ZeroEnergy();

	if (pMeshCUDA->GetMeshType() == MESH_ANTIFERROMAGNETIC) {

		//anti-ferromagnetic mesh

		DMExchangeCUDA_AFM_GetEnergy_Max <<< (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> (pMeshCUDA->cuMesh, energy, rectangle);
	}
	else {

		//ferromagnetic mesh

		DMExchangeCUDA_FM_GetEnergy_Max <<< (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> (pMeshCUDA->cuMesh, energy, rectangle);
	}

	return energy.to_cpu();
}

//////////////////////////////////////////////////////////////////////// ENERGY DENSITY DISPLAY METHODS

__global__ void DMExchangeCUDA_FM_Compute_Exchange(ManagedMeshCUDA& cuMesh, cuVEC<cuBReal>& exchange_displayVEC)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < M.linear_size()) {

		cuReal3 Hexch = cuReal3();

		if (M.is_not_empty(idx)) {

			cuBReal Ms = *cuMesh.pMs;
			cuBReal A = *cuMesh.pA;
			cuBReal D = *cuMesh.pD;
			cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs, Ms, *cuMesh.pA, A, *cuMesh.pD, D);

			if (M.is_interior(idx)) {

				//interior point : can use cheaper neu versions

				//direct exchange contribution
				Hexch = 2 * A * M.delsq_neu(idx) / ((cuBReal)MU0 * Ms * Ms);

				//Dzyaloshinskii-Moriya exchange contribution

				//Hdm, ex = -2D / (mu0*Ms) * curl m
				Hexch += -2 * D * M.curl_neu(idx) / ((cuBReal)MU0 * Ms * Ms);
			}
			else {

				//Non-homogeneous Neumann boundary conditions apply when using DMI. Required to ensure Brown's condition is fulfilled, i.e. m x h -> 0 when relaxing.
				cuReal3 bnd_dm_dx = (D / (2 * A)) * cuReal3(0, -M[idx].z, M[idx].y);
				cuReal3 bnd_dm_dy = (D / (2 * A)) * cuReal3(M[idx].z, 0, -M[idx].x);
				cuReal3 bnd_dm_dz = (D / (2 * A)) * cuReal3(-M[idx].y, M[idx].x, 0);
				cuReal33 bnd_nneu = cuReal33(bnd_dm_dx, bnd_dm_dy, bnd_dm_dz);

				//direct exchange contribution
				Hexch = 2 * A * M.delsq_nneu(idx, bnd_nneu) / ((cuBReal)MU0 * Ms * Ms);

				//Dzyaloshinskii-Moriya exchange contribution

				//Hdm, ex = -2D / (mu0*Ms) * curl m
				Hexch += -2 * D * M.curl_nneu(idx, bnd_nneu) / ((cuBReal)MU0 * Ms * Ms);
			}
		}

		exchange_displayVEC[idx] = -(cuBReal)MU0 * (M[idx] * Hexch) / 2;
	}
}

__global__ void DMExchangeCUDA_AFM_Compute_Exchange(ManagedMeshCUDA& cuMesh, cuVEC<cuBReal>& exchange_displayVEC)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC_VC<cuReal3>& M2 = *cuMesh.pM2;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < M.linear_size()) {

		cuReal3 Hexch = cuReal3();
		cuReal3 Hexch2 = cuReal3();

		if (M.is_not_empty(idx)) {

			cuReal2 Ms_AFM = *cuMesh.pMs_AFM;
			cuReal2 A_AFM = *cuMesh.pA_AFM;
			cuReal2 Ah = *cuMesh.pAh;
			cuReal2 Anh = *cuMesh.pAnh;
			cuReal2 D_AFM = *cuMesh.pD_AFM;
			cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs_AFM, Ms_AFM, *cuMesh.pA_AFM, A_AFM, *cuMesh.pAh, Ah, *cuMesh.pAnh, Anh, *cuMesh.pD_AFM, D_AFM);

			if (M.is_interior(idx)) {

				//interior point : can use cheaper neu versions

				//1. direct exchange contribution + AFM contribution
				cuReal3 delsq_M_A = M.delsq_neu(idx);
				cuReal3 delsq_M_B = M2.delsq_neu(idx);

				cuReal2 Mmag = cuReal2(M[idx].norm(), M2[idx].norm());

				Hexch = 2 * A_AFM.i * delsq_M_A / ((cuBReal)MU0 * Ms_AFM.i * Ms_AFM.i) + (-4 * Ah.i * (M[idx] ^ (M[idx] ^ M2[idx])) / (Mmag.i*Mmag.i) + Anh.i * delsq_M_B) / ((cuBReal)MU0*Ms_AFM.i*Ms_AFM.j);
				Hexch2 = 2 * A_AFM.j * delsq_M_B / ((cuBReal)MU0 * Ms_AFM.j * Ms_AFM.j) + (-4 * Ah.j * (M2[idx] ^ (M2[idx] ^ M[idx])) / (Mmag.j*Mmag.j) + Anh.j * delsq_M_A) / ((cuBReal)MU0*Ms_AFM.i*Ms_AFM.j);

				//Dzyaloshinskii-Moriya exchange contribution

				//Hdm, ex = -2D / (mu0*Ms) * curl m
				Hexch += -2 * D_AFM.i * M.curl_neu(idx) / ((cuBReal)MU0 * Ms_AFM.i * Ms_AFM.i);
				Hexch2 += -2 * D_AFM.j * M2.curl_neu(idx) / ((cuBReal)MU0 * Ms_AFM.j * Ms_AFM.j);
			}
			else {

				//Non-homogeneous Neumann boundary conditions apply when using DMI. Required to ensure Brown's condition is fulfilled, i.e. m x h -> 0 when relaxing.
				cuReal3 bnd_dm_dx = (D_AFM.i / (2 * A_AFM.i)) * cuReal3(0, -M[idx].z, M[idx].y);
				cuReal3 bnd_dm_dy = (D_AFM.i / (2 * A_AFM.i)) * cuReal3(M[idx].z, 0, -M[idx].x);
				cuReal3 bnd_dm_dz = (D_AFM.i / (2 * A_AFM.i)) * cuReal3(-M[idx].y, M[idx].x, 0);
				cuReal33 bndA_nneu = cuReal33(bnd_dm_dx, bnd_dm_dy, bnd_dm_dz);

				bnd_dm_dx = (D_AFM.j / (2 * A_AFM.j)) * cuReal3(0, -M2[idx].z, M2[idx].y);
				bnd_dm_dy = (D_AFM.j / (2 * A_AFM.j)) * cuReal3(M2[idx].z, 0, -M2[idx].x);
				bnd_dm_dz = (D_AFM.j / (2 * A_AFM.j)) * cuReal3(-M2[idx].y, M2[idx].x, 0);
				cuReal33 bndB_nneu = cuReal33(bnd_dm_dx, bnd_dm_dy, bnd_dm_dz);

				cuReal3 delsq_M_A = M.delsq_nneu(idx, bndA_nneu);
				cuReal3 delsq_M_B = M2.delsq_nneu(idx, bndB_nneu);

				cuReal2 Mmag = cuReal2(M[idx].norm(), M2[idx].norm());

				//1. direct exchange contribution + AFM contribution

				//cells marked with cmbnd are calculated using exchange coupling to other ferromagnetic meshes - see below; the delsq_nneu evaluates to zero in the CMBND coupling direction.
				Hexch = 2 * A_AFM.i * delsq_M_A / ((cuBReal)MU0 * Ms_AFM.i * Ms_AFM.i) + (-4 * Ah.i * (M[idx] ^ (M[idx] ^ M2[idx])) / (Mmag.i*Mmag.i) + Anh.i * delsq_M_B) / ((cuBReal)MU0*Ms_AFM.i*Ms_AFM.j);
				Hexch2 = 2 * A_AFM.j * delsq_M_B / ((cuBReal)MU0 * Ms_AFM.j * Ms_AFM.j) + (-4 * Ah.j * (M2[idx] ^ (M2[idx] ^ M[idx])) / (Mmag.j*Mmag.j) + Anh.j * delsq_M_A) / ((cuBReal)MU0*Ms_AFM.i*Ms_AFM.j);

				//2. Dzyaloshinskii-Moriya exchange contribution

				//Hdm, ex = -2D / (mu0*Ms) * curl m
				//For cmbnd cells curl_nneu does not evaluate to zero in the CMBND coupling direction, but sided differentials are used - when setting values at CMBND cells for exchange coupled meshes must correct for this.
				Hexch += -2 * D_AFM.i * M.curl_nneu(idx, bndA_nneu) / ((cuBReal)MU0 * Ms_AFM.i * Ms_AFM.i);
				Hexch2 += -2 * D_AFM.j * M2.curl_nneu(idx, bndB_nneu) / ((cuBReal)MU0 * Ms_AFM.j * Ms_AFM.j);
			}
		}

		exchange_displayVEC[idx] = -(cuBReal)MU0 * (M[idx] * Hexch + M2[idx] * Hexch2) / 4;
	}
}

void DMExchangeCUDA::Compute_ExchangeCUDA(void)
{
	exchange_displayVEC()->resize(pMeshCUDA->h, pMeshCUDA->meshRect);

	if (pMeshCUDA->GetMeshType() == MESH_ANTIFERROMAGNETIC) {

		//anti-ferromagnetic mesh

		DMExchangeCUDA_AFM_Compute_Exchange <<< (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> (pMeshCUDA->cuMesh, exchange_displayVEC);
	}
	else {

		//ferromagnetic mesh

		DMExchangeCUDA_FM_Compute_Exchange <<< (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> (pMeshCUDA->cuMesh, exchange_displayVEC);
	}
}

#endif

#endif