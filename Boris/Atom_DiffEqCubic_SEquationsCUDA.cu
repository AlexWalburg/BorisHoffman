#include "hip/hip_runtime.h"
#include "Atom_DiffEqCubicCUDA.h"

#if COMPILECUDA == 1
#ifdef MESH_COMPILATION_ATOM_CUBIC

#include "BorisCUDALib.h"
#include "BorisCUDALib.cuh"

#include "Atom_MeshParamsControlCUDA.h"

//---------------------------------------- OTHER CALCULATION METHODS : GENERATE THERMAL cuVECs

//----------------------------------------

__global__ void GenerateThermalField_Kernel(cuBorisRand& prng, ManagedAtom_DiffEqCubicCUDA& cuaDiffEq, ManagedAtom_MeshCUDA& cuaMesh, cuBReal& deltaT)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < cuaDiffEq.pH_Thermal->linear_size()) {

		cuReal3 position = cuaDiffEq.pH_Thermal->cellidx_to_position(idx);

		if (cuaMesh.pM1->is_not_empty(position) && !cuaMesh.pM1->is_skipcell(position)) {

			cuReal3 h = cuaDiffEq.pH_Thermal->h;

			cuBReal Temperature;

			if (cuaMesh.pTemp->linear_size()) {

				//get temperature at centre of idx M cell
				Temperature = (*cuaMesh.pTemp)[position];
			}
			else Temperature = (*cuaMesh.pbase_temperature);

			cuBReal mu_s = *cuaMesh.pmu_s;
			cuaMesh.update_parameters_mcoarse(idx, *cuaMesh.pmu_s, mu_s);

			//do not include any damping here - this will be included in the stochastic equations
			cuBReal Hth_const = sqrt(2 * (cuBReal)BOLTZMANN * Temperature / ((cuBReal)MUB_MU0 * GAMMA * mu_s * deltaT));
			
			(*cuaDiffEq.pH_Thermal)[idx] = Hth_const * cuReal3(prng.rand_gauss(0, 1), prng.rand_gauss(0, 1), prng.rand_gauss(0, 1));
		}
	}
}

//called when using stochastic equations
void Atom_DifferentialEquationCubicCUDA::GenerateThermalField_CUDA(cu_obj<cuBReal>& deltaT)
{
	GenerateThermalField_Kernel <<< (paMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> (prng, cuaDiffEq, paMeshCUDA->cuaMesh, deltaT);
}

#endif
#endif