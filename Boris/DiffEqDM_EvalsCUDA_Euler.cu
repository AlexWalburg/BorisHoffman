#include "hip/hip_runtime.h"
#include "DiffEqDMCUDA.h"

#if COMPILECUDA == 1
#ifdef ODE_EVAL_COMPILATION_EULER
#ifdef MESH_COMPILATION_DIAMAGNETIC

#include "MeshParamsControlCUDA.h"

//defines evaluation methods kernel launchers

#include "BorisCUDALib.cuh"

//----------------------------------------- EVALUATIONS: Euler

__global__ void RunEuler_Kernel(ManagedDiffEqDMCUDA& cuDiffEq, ManagedMeshCUDA& cuMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < cuMesh.pM->linear_size()) {

		if (cuMesh.pM->is_not_empty(idx)) {

			//Save current magnetization in case we need to restore it due to evaluations in other meshes
			(*cuDiffEq.psM1)[idx] = (*cuMesh.pM)[idx];

			//Set M from diamagnetic susceptibility
			(*cuMesh.pM)[idx] = (cuDiffEq.*(cuDiffEq.pODEFunc))(idx);
		}
	}
}

//----------------------------------------- DifferentialEquationCUDA Launchers

//EULER

void DifferentialEquationDMCUDA::RunEuler(bool calculate_mxh, bool calculate_dmdt)
{
	RunEuler_Kernel << < (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >> > (cuDiffEq, pMeshCUDA->cuMesh);
}

#endif
#endif
#endif