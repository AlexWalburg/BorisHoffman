#include "hip/hip_runtime.h"
#include "AnisotropyCubiCUDA.h"

#if COMPILECUDA == 1

#ifdef MODULE_COMPILATION_ANICUBI

#include "BorisCUDALib.cuh"

#include "MeshCUDA.h"
#include "MeshParamsControlCUDA.h"
#include "MeshDefs.h"

__global__ void Anisotropy_CubicCUDA_FM_UpdateField(ManagedMeshCUDA& cuMesh, cuBReal& energy, bool do_reduction)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC<cuReal3>& Heff = *cuMesh.pHeff;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal energy_ = 0.0;

	if (idx < Heff.linear_size()) {

		cuReal3 Heff_value = cuReal3();

		if (M.is_not_empty(idx)) {

			cuBReal Ms = *cuMesh.pMs;
			cuBReal K1 = *cuMesh.pK1;
			cuBReal K2 = *cuMesh.pK2;
			cuReal3 mcanis_ea1 = *cuMesh.pmcanis_ea1;
			cuReal3 mcanis_ea2 = *cuMesh.pmcanis_ea2;

			cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs, Ms, *cuMesh.pK1, K1, *cuMesh.pK2, K2, *cuMesh.pmcanis_ea1, mcanis_ea1, *cuMesh.pmcanis_ea2, mcanis_ea2);

			//vector product of ea1 and ea2 : the third orthogonal axis
			cuReal3 mcanis_ea3 = mcanis_ea1 ^ mcanis_ea2;

			//calculate m.ea1, m.ea2 and m.ea3 dot products
			cuBReal d1 = (M[idx] * mcanis_ea1) / Ms;
			cuBReal d2 = (M[idx] * mcanis_ea2) / Ms;
			cuBReal d3 = (M[idx] * mcanis_ea3) / Ms;

			//terms for K1 contribution
			cuBReal a1 = d1 * (d2*d2 + d3 * d3);
			cuBReal a2 = d2 * (d1*d1 + d3 * d3);
			cuBReal a3 = d3 * (d1*d1 + d2 * d2);

			//terms for K2 contribution
			cuBReal d123 = d1 * d2*d3;

			cuBReal b1 = d123 * d2*d3;
			cuBReal b2 = d123 * d1*d3;
			cuBReal b3 = d123 * d1*d2;

			//update effective field with the anisotropy field
			Heff_value = cuReal3(
				(-2 * K1 / ((cuBReal)MU0*Ms)) * (mcanis_ea1.i * a1 + mcanis_ea2.i * a2 + mcanis_ea3.i * a3)
				+ (-2 * K2 / ((cuBReal)MU0*Ms)) * (mcanis_ea1.i * b1 + mcanis_ea2.i * b2 + mcanis_ea3.i * b3),

				(-2 * K1 / ((cuBReal)MU0*Ms)) * (mcanis_ea1.j * a1 + mcanis_ea2.j * a2 + mcanis_ea3.j * a3)
				+ (-2 * K2 / ((cuBReal)MU0*Ms)) * (mcanis_ea1.j * b1 + mcanis_ea2.j * b2 + mcanis_ea3.j * b3),

				(-2 * K1 / ((cuBReal)MU0*Ms)) * (mcanis_ea1.k * a1 + mcanis_ea2.k * a2 + mcanis_ea3.k * a3)
				+ (-2 * K2 / ((cuBReal)MU0*Ms)) * (mcanis_ea1.k * b1 + mcanis_ea2.k * b2 + mcanis_ea3.k * b3)
			);

			if (do_reduction) {

				//update energy (E/V)		
				int non_empty_cells = M.get_nonempty_cells();
				if (non_empty_cells) energy_ = (K1 * (d1*d1*d2*d2 + d1 * d1*d3*d3 + d2 * d2*d3*d3) + K2 * d123*d123) / non_empty_cells;
			}
		}

		Heff[idx] += Heff_value;
	}

	if (do_reduction) reduction_sum(0, 1, &energy_, energy);
}

__global__ void Anisotropy_CubicCUDA_AFM_UpdateField(ManagedMeshCUDA& cuMesh, cuBReal& energy, bool do_reduction)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC<cuReal3>& Heff = *cuMesh.pHeff;

	cuVEC_VC<cuReal3>& M2 = *cuMesh.pM2;
	cuVEC<cuReal3>& Heff2 = *cuMesh.pHeff2;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal energy_ = 0.0;

	if (idx < Heff.linear_size()) {

		cuReal3 Heff_value = cuReal3();
		cuReal3 Heff2_value = cuReal3();

		if (M.is_not_empty(idx)) {

			cuReal2 Ms_AFM = *cuMesh.pMs_AFM;
			cuReal2 K1_AFM = *cuMesh.pK1_AFM;
			cuReal2 K2_AFM = *cuMesh.pK2_AFM;
			cuReal3 mcanis_ea1 = *cuMesh.pmcanis_ea1;
			cuReal3 mcanis_ea2 = *cuMesh.pmcanis_ea2;

			cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs_AFM, Ms_AFM, *cuMesh.pK1_AFM, K1_AFM, *cuMesh.pK2_AFM, K2_AFM, *cuMesh.pmcanis_ea1, mcanis_ea1, *cuMesh.pmcanis_ea2, mcanis_ea2);

			//vector product of ea1 and ea2 : the third orthogonal axis
			cuReal3 mcanis_ea3 = mcanis_ea1 ^ mcanis_ea2;

			//calculate m.ea1, m.ea2 and m.ea3 dot products
			cuBReal d1 = (M[idx] * mcanis_ea1) / Ms_AFM.i;
			cuBReal d2 = (M[idx] * mcanis_ea2) / Ms_AFM.i;
			cuBReal d3 = (M[idx] * mcanis_ea3) / Ms_AFM.i;

			//terms for K1 contribution
			cuBReal a1 = d1 * (d2*d2 + d3 * d3);
			cuBReal a2 = d2 * (d1*d1 + d3 * d3);
			cuBReal a3 = d3 * (d1*d1 + d2 * d2);

			//terms for K2 contribution
			cuBReal d123 = d1*d2*d3;

			cuBReal b1 = d123 * d2*d3;
			cuBReal b2 = d123 * d1*d3;
			cuBReal b3 = d123 * d1*d2;

			//update effective field with the anisotropy field
			Heff_value = cuReal3(
				(-2 * K1_AFM.i / ((cuBReal)MU0*Ms_AFM.i)) * (mcanis_ea1.i * a1 + mcanis_ea2.i * a2 + mcanis_ea3.i * a3)
				+ (-2 * K2_AFM.i / ((cuBReal)MU0*Ms_AFM.i)) * (mcanis_ea1.i * b1 + mcanis_ea2.i * b2 + mcanis_ea3.i * b3),

				(-2 * K1_AFM.i / ((cuBReal)MU0*Ms_AFM.i)) * (mcanis_ea1.j * a1 + mcanis_ea2.j * a2 + mcanis_ea3.j * a3)
				+ (-2 * K2_AFM.i / ((cuBReal)MU0*Ms_AFM.i)) * (mcanis_ea1.j * b1 + mcanis_ea2.j * b2 + mcanis_ea3.j * b3),

				(-2 * K1_AFM.i / ((cuBReal)MU0*Ms_AFM.i)) * (mcanis_ea1.k * a1 + mcanis_ea2.k * a2 + mcanis_ea3.k * a3)
				+ (-2 * K2_AFM.i / ((cuBReal)MU0*Ms_AFM.i)) * (mcanis_ea1.k * b1 + mcanis_ea2.k * b2 + mcanis_ea3.k * b3)
			);

			//same thing for sub-lattice B

			cuBReal d1B = (M2[idx] * mcanis_ea1) / Ms_AFM.j;
			cuBReal d2B = (M2[idx] * mcanis_ea2) / Ms_AFM.j;
			cuBReal d3B = (M2[idx] * mcanis_ea3) / Ms_AFM.j;

			cuBReal a1B = d1B * (d2B*d2B + d3B*d3B);
			cuBReal a2B = d2B * (d1B*d1B + d3B*d3B);
			cuBReal a3B = d3B * (d1B*d1B + d2B*d2B);

			cuBReal d123B = d1B*d2B*d3B;

			cuBReal b1B = d123B*d2B*d3B;
			cuBReal b2B = d123B*d1B*d3B;
			cuBReal b3B = d123B*d1B*d2B;

			Heff2_value = cuReal3(
				(-2 * K1_AFM.j / ((cuBReal)MU0*Ms_AFM.j)) * (mcanis_ea1.i * a1B + mcanis_ea2.i * a2B + mcanis_ea3.i * a3B)
				+ (-2 * K2_AFM.j / ((cuBReal)MU0*Ms_AFM.j)) * (mcanis_ea1.i * b1B + mcanis_ea2.i * b2B + mcanis_ea3.i * b3B),

				(-2 * K1_AFM.j / ((cuBReal)MU0*Ms_AFM.j)) * (mcanis_ea1.j * a1B + mcanis_ea2.j * a2B + mcanis_ea3.j * a3B)
				+ (-2 * K2_AFM.j / ((cuBReal)MU0*Ms_AFM.j)) * (mcanis_ea1.j * b1B + mcanis_ea2.j * b2B + mcanis_ea3.j * b3B),

				(-2 * K1_AFM.j / ((cuBReal)MU0*Ms_AFM.j)) * (mcanis_ea1.k * a1B + mcanis_ea2.k * a2B + mcanis_ea3.k * a3B)
				+ (-2 * K2_AFM.j / ((cuBReal)MU0*Ms_AFM.j)) * (mcanis_ea1.k * b1B + mcanis_ea2.k * b2B + mcanis_ea3.k * b3B)
			);

			if (do_reduction) {

				//update energy (E/V)		
				int non_empty_cells = M.get_nonempty_cells();
				if (non_empty_cells) energy_ = ((K1_AFM.i * (d1*d1*d2*d2 + d1*d1*d3*d3 + d2*d2*d3*d3) + K2_AFM.i * d123*d123) + (K1_AFM.j * (d1B*d1B*d2B*d2B + d1B*d1B*d3B*d3B + d2B*d2B*d3B*d3B) + K2_AFM.j * d123B*d123B)) / (2*non_empty_cells);
			}
		}

		Heff[idx] += Heff_value;
		Heff2[idx] += Heff2_value;
	}

	if (do_reduction) reduction_sum(0, 1, &energy_, energy);
}

//----------------------- UpdateField LAUNCHER

void Anisotropy_CubicCUDA::UpdateField(void)
{
	if (pMeshCUDA->GetMeshType() == MESH_ANTIFERROMAGNETIC) {

		//anti-ferromagnetic mesh

		if (pMeshCUDA->CurrentTimeStepSolved()) {

			ZeroEnergy();

			Anisotropy_CubicCUDA_AFM_UpdateField << < (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >> > (pMeshCUDA->cuMesh, energy, true);
		}
		else {

			Anisotropy_CubicCUDA_AFM_UpdateField << < (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >> > (pMeshCUDA->cuMesh, energy, false);
		}
	}
	else {

		//ferromagnetic mesh

		if (pMeshCUDA->CurrentTimeStepSolved()) {

			ZeroEnergy();

			Anisotropy_CubicCUDA_FM_UpdateField << < (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >> > (pMeshCUDA->cuMesh, energy, true);
		}
		else {

			Anisotropy_CubicCUDA_FM_UpdateField << < (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >> > (pMeshCUDA->cuMesh, energy, false);
		}
	}
}

//-------------------Energy density methods

__global__ void Anisotropy_CubicCUDA_FM_GetEnergy(ManagedMeshCUDA& cuMesh, cuBReal& energy, size_t& points_count, cuRect avRect)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal energy_ = 0.0;

	bool include_in_reduction = false;

	if (idx < M.linear_size()) {

		if (M.is_not_empty(idx) && avRect.contains(M.cellidx_to_position(idx))) {

			cuBReal Ms = *cuMesh.pMs;
			cuBReal K1 = *cuMesh.pK1;
			cuBReal K2 = *cuMesh.pK2;
			cuReal3 mcanis_ea1 = *cuMesh.pmcanis_ea1;
			cuReal3 mcanis_ea2 = *cuMesh.pmcanis_ea2;

			cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs, Ms, *cuMesh.pK1, K1, *cuMesh.pK2, K2, *cuMesh.pmcanis_ea1, mcanis_ea1, *cuMesh.pmcanis_ea2, mcanis_ea2);

			//vector product of ea1 and ea2 : the third orthogonal axis
			cuReal3 mcanis_ea3 = mcanis_ea1 ^ mcanis_ea2;

			//calculate m.ea1, m.ea2 and m.ea3 dot products
			cuBReal d1 = (M[idx] * mcanis_ea1) / Ms;
			cuBReal d2 = (M[idx] * mcanis_ea2) / Ms;
			cuBReal d3 = (M[idx] * mcanis_ea3) / Ms;

			//terms for K2 contribution
			cuBReal d123 = d1*d2*d3;

			energy_ = (K1 * (d1*d1*d2*d2 + d1*d1*d3*d3 + d2*d2*d3*d3) + K2 * d123*d123);
			include_in_reduction = true;
		}
	}

	reduction_avg(0, 1, &energy_, energy, points_count, include_in_reduction);
}

__global__ void Anisotropy_CubicCUDA_AFM_GetEnergy(ManagedMeshCUDA& cuMesh, cuBReal& energy, size_t& points_count, cuRect avRect)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC_VC<cuReal3>& M2 = *cuMesh.pM2;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal energy_ = 0.0;

	bool include_in_reduction = false;

	if (idx < M.linear_size()) {

		if (M.is_not_empty(idx) && avRect.contains(M.cellidx_to_position(idx))) {

			cuReal2 Ms_AFM = *cuMesh.pMs_AFM;
			cuReal2 K1_AFM = *cuMesh.pK1_AFM;
			cuReal2 K2_AFM = *cuMesh.pK2_AFM;
			cuReal3 mcanis_ea1 = *cuMesh.pmcanis_ea1;
			cuReal3 mcanis_ea2 = *cuMesh.pmcanis_ea2;

			cuMesh.update_parameters_mcoarse(idx, *cuMesh.pMs_AFM, Ms_AFM, *cuMesh.pK1_AFM, K1_AFM, *cuMesh.pK2_AFM, K2_AFM, *cuMesh.pmcanis_ea1, mcanis_ea1, *cuMesh.pmcanis_ea2, mcanis_ea2);

			//vector product of ea1 and ea2 : the third orthogonal axis
			cuReal3 mcanis_ea3 = mcanis_ea1 ^ mcanis_ea2;

			//calculate m.ea1, m.ea2 and m.ea3 dot products
			cuBReal d1 = (M[idx] * mcanis_ea1) / Ms_AFM.i;
			cuBReal d2 = (M[idx] * mcanis_ea2) / Ms_AFM.i;
			cuBReal d3 = (M[idx] * mcanis_ea3) / Ms_AFM.i;

			//terms for K2 contribution
			cuBReal d123 = d1*d2*d3;

			//same thing for sub-lattice B

			cuBReal d1B = (M2[idx] * mcanis_ea1) / Ms_AFM.j;
			cuBReal d2B = (M2[idx] * mcanis_ea2) / Ms_AFM.j;
			cuBReal d3B = (M2[idx] * mcanis_ea3) / Ms_AFM.j;

			cuBReal d123B = d1B*d2B*d3B;

			energy_ = ((K1_AFM.i * (d1*d1*d2*d2 + d1*d1*d3*d3 + d2*d2*d3*d3) + K2_AFM.i * d123*d123) + (K1_AFM.j * (d1B*d1B*d2B*d2B + d1B * d1B*d3B*d3B + d2B * d2B*d3B*d3B) + K2_AFM.j * d123B*d123B)) / 2;
			include_in_reduction = true;
		}
	}

	reduction_avg(0, 1, &energy_, energy, points_count, include_in_reduction);
}

cuBReal Anisotropy_CubicCUDA::GetEnergyDensity(cuRect avRect)
{
	ZeroEnergy();

	if (pMeshCUDA->GetMeshType() == MESH_ANTIFERROMAGNETIC) {

		//anti-ferromagnetic mesh

		Anisotropy_CubicCUDA_AFM_GetEnergy <<< (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> (pMeshCUDA->cuMesh, energy, points_count, avRect);
	}
	else {

		//ferromagnetic mesh

		Anisotropy_CubicCUDA_FM_GetEnergy <<< (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> (pMeshCUDA->cuMesh, energy, points_count, avRect);
	}

	size_t points_count_cpu = points_count.to_cpu();

	if (points_count_cpu) return energy.to_cpu() / points_count_cpu;
	else return 0.0;
}

#endif

#endif