#include "hip/hip_runtime.h"
#include "DiffEqDMCUDA.h"

#if COMPILECUDA == 1
#ifdef ODE_EVAL_COMPILATION_RKDP
#ifdef MESH_COMPILATION_DIAMAGNETIC

#include "MeshParamsControlCUDA.h"

//defines evaluation methods kernel launchers

#include "BorisCUDALib.cuh"

//----------------------------------------- EVALUATIONS : RKDP54

__global__ void RunRKDP54_Step0_Kernel(ManagedDiffEqDMCUDA& cuDiffEq, ManagedMeshCUDA& cuMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < cuMesh.pM->linear_size()) {

		if (cuMesh.pM->is_not_empty(idx)) {

			//Set M from diamagnetic susceptibility
			(*cuMesh.pM)[idx] = (cuDiffEq.*(cuDiffEq.pODEFunc))(idx);
		}
	}
}

__global__ void RunRKDP54_Step0_Advance_Kernel(ManagedDiffEqDMCUDA& cuDiffEq, ManagedMeshCUDA& cuMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < cuMesh.pM->linear_size()) {

		if (cuMesh.pM->is_not_empty(idx)) {

			//Save current magnetization in case we need to restore it due to evaluations in other meshes
			(*cuDiffEq.psM1)[idx] = (*cuMesh.pM)[idx];

			//Set M from diamagnetic susceptibility
			(*cuMesh.pM)[idx] = (cuDiffEq.*(cuDiffEq.pODEFunc))(idx);
		}
	}
}

__global__ void RunRKDP54_Step1_Kernel(ManagedDiffEqDMCUDA& cuDiffEq, ManagedMeshCUDA& cuMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < cuMesh.pM->linear_size()) {

		if (cuMesh.pM->is_not_empty(idx)) {

			//Set M from diamagnetic susceptibility
			(*cuMesh.pM)[idx] = (cuDiffEq.*(cuDiffEq.pODEFunc))(idx);
		}
	}
}

__global__ void RunRKDP54_Step2_Kernel(ManagedDiffEqDMCUDA& cuDiffEq, ManagedMeshCUDA& cuMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < cuMesh.pM->linear_size()) {

		if (cuMesh.pM->is_not_empty(idx)) {

			//Set M from diamagnetic susceptibility
			(*cuMesh.pM)[idx] = (cuDiffEq.*(cuDiffEq.pODEFunc))(idx);
		}
	}
}

__global__ void RunRKDP54_Step3_Kernel(ManagedDiffEqDMCUDA& cuDiffEq, ManagedMeshCUDA& cuMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < cuMesh.pM->linear_size()) {

		if (cuMesh.pM->is_not_empty(idx)) {

			//Set M from diamagnetic susceptibility
			(*cuMesh.pM)[idx] = (cuDiffEq.*(cuDiffEq.pODEFunc))(idx);
		}
	}
}

__global__ void RunRKDP54_Step4_Kernel(ManagedDiffEqDMCUDA& cuDiffEq, ManagedMeshCUDA& cuMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < cuMesh.pM->linear_size()) {

		if (cuMesh.pM->is_not_empty(idx)) {

			//Set M from diamagnetic susceptibility
			(*cuMesh.pM)[idx] = (cuDiffEq.*(cuDiffEq.pODEFunc))(idx);
		}
	}
}

__global__ void RunRKDP54_Step5_Kernel(ManagedDiffEqDMCUDA& cuDiffEq, ManagedMeshCUDA& cuMesh)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < cuMesh.pM->linear_size()) {

		if (cuMesh.pM->is_not_empty(idx)) {

			//Set M from diamagnetic susceptibility
			(*cuMesh.pM)[idx] = (cuDiffEq.*(cuDiffEq.pODEFunc))(idx);
		}
	}
}

//----------------------------------------- DifferentialEquationCUDA Launchers

//RUNGE KUTTA DORMAND-PRINCE

void DifferentialEquationDMCUDA::RunRKDP54_Step0_NoAdvance(bool calculate_mxh)
{
	RunRKDP54_Step0_Kernel <<< (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> (cuDiffEq, pMeshCUDA->cuMesh);
}

void DifferentialEquationDMCUDA::RunRKDP54(int step, bool calculate_mxh, bool calculate_dmdt)
{
	switch (step) {

	case 0:

		RunRKDP54_Step0_Advance_Kernel << < (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >> > (cuDiffEq, pMeshCUDA->cuMesh);

		break;

	case 1:

		RunRKDP54_Step1_Kernel << < (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >> > (cuDiffEq, pMeshCUDA->cuMesh);

		break;

	case 2:

		RunRKDP54_Step2_Kernel << < (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >> > (cuDiffEq, pMeshCUDA->cuMesh);

		break;

	case 3:

		RunRKDP54_Step3_Kernel << < (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >> > (cuDiffEq, pMeshCUDA->cuMesh);

		break;

	case 4:

		RunRKDP54_Step4_Kernel << < (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >> > (cuDiffEq, pMeshCUDA->cuMesh);

		break;

	case 5:

		RunRKDP54_Step5_Kernel << < (pMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >> > (cuDiffEq, pMeshCUDA->cuMesh);

		break;
	}
}

#endif
#endif
#endif