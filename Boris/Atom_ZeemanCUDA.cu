#include "hip/hip_runtime.h"
#include "Atom_ZeemanCUDA.h"

#if COMPILECUDA == 1

#if defined(MODULE_COMPILATION_ZEEMAN) && ATOMISTIC == 1

#include "BorisCUDALib.cuh"

#include "MeshDefs.h"

#include "Atom_MeshCUDA.h"
#include "Atom_MeshParamsControlCUDA.h"

__global__ void Atom_ZeemanCUDA_UpdateField_Cubic(ManagedAtom_MeshCUDA& cuMesh, cuReal3& Ha, cuBReal& energy, bool do_reduction)
{
	cuVEC_VC<cuReal3>& M1 = *cuMesh.pM1;
	cuVEC<cuReal3>& Heff1 = *cuMesh.pHeff1;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal energy_ = 0.0;

	if (idx < Heff1.linear_size()) {

		cuBReal cHA = *cuMesh.pcHA;
		cuMesh.update_parameters_mcoarse(idx, *cuMesh.pcHA, cHA);

		Heff1[idx] += (cHA * Ha);

		if (do_reduction) {

			//energy density
			int non_empty_cells = M1.get_nonempty_cells();
			if (non_empty_cells) energy_ = -(cuBReal)MUB * M1[idx] * (cuBReal)MU0 * (cHA * Ha) / (non_empty_cells * M1.h.dim());
		}
	}

	if (do_reduction) reduction_sum(0, 1, &energy_, energy);
}

__global__ void Atom_ZeemanCUDA_UpdateField_Equation_Cubic(
	ManagedAtom_MeshCUDA& cuMesh,
	ManagedFunctionCUDA<cuBReal, cuBReal, cuBReal, cuBReal>& H_equation_x,
	ManagedFunctionCUDA<cuBReal, cuBReal, cuBReal, cuBReal>& H_equation_y,
	ManagedFunctionCUDA<cuBReal, cuBReal, cuBReal, cuBReal>& H_equation_z,
	cuBReal time,
	cuBReal& energy, bool do_reduction)
{
	cuVEC_VC<cuReal3>& M1 = *cuMesh.pM1;
	cuVEC<cuReal3>& Heff1 = *cuMesh.pHeff1;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal energy_ = 0.0;

	if (idx < Heff1.linear_size()) {

		cuBReal cHA = *cuMesh.pcHA;
		cuMesh.update_parameters_mcoarse(idx, *cuMesh.pcHA, cHA);

		cuReal3 relpos = M1.cellidx_to_position(idx);
		cuReal3 H = cuReal3(
			H_equation_x.evaluate(relpos.x, relpos.y, relpos.z, time),
			H_equation_y.evaluate(relpos.x, relpos.y, relpos.z, time),
			H_equation_z.evaluate(relpos.x, relpos.y, relpos.z, time));

		Heff1[idx] += (cHA * H);

		if (do_reduction) {

			//energy density
			int non_empty_cells = M1.get_nonempty_cells();
			if (non_empty_cells) energy_ = -(cuBReal)MUB * M1[idx] * (cuBReal)MU0 * (cHA * H) / (non_empty_cells * M1.h.dim());
		}
	}

	if (do_reduction) reduction_sum(0, 1, &energy_, energy);
}

//----------------------- UpdateField LAUNCHER

void Atom_ZeemanCUDA::UpdateField(void)
{
	/////////////////////////////////////////
	// Fixed set field
	/////////////////////////////////////////

	if (!H_equation.is_set()) {

		if (paMeshCUDA->GetMeshType() == MESH_ATOM_CUBIC) {

			if (paMeshCUDA->CurrentTimeStepSolved()) {

				ZeroEnergy();

				Atom_ZeemanCUDA_UpdateField_Cubic <<< (paMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> (paMeshCUDA->cuaMesh, Ha, energy, true);
			}
			else Atom_ZeemanCUDA_UpdateField_Cubic <<< (paMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> (paMeshCUDA->cuaMesh, Ha, energy, false);
		}
	}

	/////////////////////////////////////////
	// Field set from user equation
	/////////////////////////////////////////

	else {

		if (paMeshCUDA->GetMeshType() == MESH_ATOM_CUBIC) {

			if (paMeshCUDA->CurrentTimeStepSolved()) {

				ZeroEnergy();

				Atom_ZeemanCUDA_UpdateField_Equation_Cubic <<< (paMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> (
					paMeshCUDA->cuaMesh,
					H_equation.get_x(), H_equation.get_y(), H_equation.get_z(),
					paMeshCUDA->GetStageTime(),
					energy, true);
			}
			else Atom_ZeemanCUDA_UpdateField_Equation_Cubic <<< (paMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> (
				paMeshCUDA->cuaMesh,
				H_equation.get_x(), H_equation.get_y(), H_equation.get_z(),
				paMeshCUDA->GetStageTime(),
				energy, false);
		}
	}
}

//-------------------Energy density methods

__global__ void Atom_ZeemanCUDA_GetEnergy_Cubic(ManagedAtom_MeshCUDA& cuMesh, cuReal3& Ha, cuBReal& energy, size_t& points_count, cuRect avRect)
{
	cuVEC_VC<cuReal3>& M1 = *cuMesh.pM1;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal energy_ = 0.0;

	bool include_in_reduction = false;

	if (idx < M1.linear_size()) {

		if (M1.is_not_empty(idx) && avRect.contains(M1.cellidx_to_position(idx))) {

			cuBReal cHA = *cuMesh.pcHA;
			cuMesh.update_parameters_mcoarse(idx, *cuMesh.pcHA, cHA);

			//energy, not energy density : will convert to energy density later
			energy_ = -(cuBReal)MUB * M1[idx] * (cuBReal)MU0 * (cHA * Ha);
			include_in_reduction = true;
		}
	}

	reduction_avg(0, 1, &energy_, energy, points_count, include_in_reduction);
}

__global__ void Atom_ZeemanCUDA_GetEnergy_Equation_Cubic(
	ManagedAtom_MeshCUDA& cuMesh,
	ManagedFunctionCUDA<cuBReal, cuBReal, cuBReal, cuBReal>& H_equation_x,
	ManagedFunctionCUDA<cuBReal, cuBReal, cuBReal, cuBReal>& H_equation_y,
	ManagedFunctionCUDA<cuBReal, cuBReal, cuBReal, cuBReal>& H_equation_z,
	cuBReal time,
	cuBReal& energy, size_t& points_count, cuRect avRect)
{
	cuVEC_VC<cuReal3>& M1 = *cuMesh.pM1;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal energy_ = 0.0;

	bool include_in_reduction = false;

	if (idx < M1.linear_size()) {

		if (M1.is_not_empty(idx) && avRect.contains(M1.cellidx_to_position(idx))) {

			cuBReal cHA = *cuMesh.pcHA;
			cuMesh.update_parameters_mcoarse(idx, *cuMesh.pcHA, cHA);

			cuReal3 relpos = M1.cellidx_to_position(idx);
			cuReal3 H = cuReal3(
				H_equation_x.evaluate(relpos.x, relpos.y, relpos.z, time),
				H_equation_y.evaluate(relpos.x, relpos.y, relpos.z, time),
				H_equation_z.evaluate(relpos.x, relpos.y, relpos.z, time));

			//energy, not energy density : will convert to energy density later
			energy_ = -(cuBReal)MUB * M1[idx] * (cuBReal)MU0 * (cHA * H);
			include_in_reduction = true;
		}
	}

	reduction_avg(0, 1, &energy_, energy, points_count, include_in_reduction);
}

cuBReal Atom_ZeemanCUDA::GetEnergyDensity(cuRect avRect)
{
	ZeroEnergy();

	/////////////////////////////////////////
	// Fixed set field
	/////////////////////////////////////////

	if (!H_equation.is_set()) {

		if (paMeshCUDA->GetMeshType() == MESH_ATOM_CUBIC) {

			Atom_ZeemanCUDA_GetEnergy_Cubic <<< (paMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> (paMeshCUDA->cuaMesh, Ha, energy, points_count, avRect);
		}
	}

	/////////////////////////////////////////
	// Field set from user equation
	/////////////////////////////////////////

	else {

		if (paMeshCUDA->GetMeshType() == MESH_ATOM_CUBIC) {

			Atom_ZeemanCUDA_GetEnergy_Equation_Cubic <<< (paMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> (
				paMeshCUDA->cuaMesh,
				H_equation.get_x(), H_equation.get_y(), H_equation.get_z(),
				paMeshCUDA->GetStageTime(),
				energy, points_count, avRect);
		}
	}

	size_t points_count_cpu = points_count.to_cpu();

	if (points_count_cpu) return energy.to_cpu() / (points_count_cpu * paMeshCUDA->h.dim());
	else return 0.0;
}

//-------------------Others

BError Atom_ZeemanCUDA::SetFieldEquation(const std::vector<std::vector< std::vector<EqComp::FSPEC> >>& fspec)
{
	BError error(CLASS_STR(Atom_ZeemanCUDA));

	if (!H_equation.make_vector(fspec)) return error(BERROR_OUTOFGPUMEMORY_CRIT);

	return error;
}

#endif

#endif